#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <algorithm>
#include <stdexcept>

#include <stdint.h>
#include <cstdio>


#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


// just for compatability of half precision in AT_DISPATCH_FLOATING_TYPES_AND_HALF... program will never reach here!
 __device__ inline at::Half atomicAdd(at::Half *address, at::Half val) {
  // requires CUDA >= 10 and ARCH >= 70
  // this is very slow compared to float or __half2, never use it.
  //return atomicAdd(reinterpret_cast<__half*>(address), val);
}


template <typename T>
__host__ __device__ inline T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

template <typename T>
__device__ inline T smoothstep(T val) {
	return val*val*(3.0f - 2.0f * val);
}

template <typename T>
__device__ inline T smoothstep_derivative(T val) {
	return 6*val*(1.0f - val);
}


template <uint32_t D>
__device__ uint32_t fast_hash(const uint32_t pos_grid[D]) {
    
    // coherent type of hashing
    constexpr uint32_t primes[7] = { 1u, 2654435761u, 805459861u, 3674653429u, 2097192037u, 1434869437u, 2165219737u };

    uint32_t result = 0;
    #pragma unroll
    for (uint32_t i = 0; i < D; ++i) {
        result ^= pos_grid[i] * primes[i];
    }

    return result;
}


template <uint32_t D, uint32_t C>
__device__ uint32_t get_grid_index(const uint32_t gridtype, const uint32_t ch, const uint32_t hashmap_size, const uint32_t resolution, const uint32_t pos_grid[D]) {
    uint32_t stride = 1;
    uint32_t index = 0;

    #pragma unroll
    for (uint32_t d = 0; d < D && stride <= hashmap_size; d++) {
        index += pos_grid[d] * stride;
        stride *= resolution;
    }

    // NOTE: for NeRF, the hash is in fact not necessary. Check https://github.com/NVlabs/instant-ngp/issues/97.
    // gridtype: 0 == hash, 1 == tiled
    if (gridtype == 0 && stride > hashmap_size) {
        index = fast_hash<D>(pos_grid);
    }

    return (index % hashmap_size) * C + ch;
}


template <typename scalar_t, uint32_t D, uint32_t C>
__global__ void kernel_grid(
    const float * __restrict__ inputs, 
    const scalar_t * __restrict__ grid, 
    const int * __restrict__ offsets, 
    scalar_t * __restrict__ outputs, 
    const uint32_t B, const uint32_t L, const float S, const uint32_t H,
    scalar_t * __restrict__ dy_dx,
    const uint32_t gridtype,
    const bool align_corners,
    const uint32_t interp
) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    
    // locate
    grid += (uint32_t)offsets[level] * C;
    inputs += b * D;
    outputs += level * B * C + b * C;

    // check input range (should be in [0, 1])
    bool flag_oob = false;
    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        if (inputs[d] < 0 || inputs[d] > 1) {
            flag_oob = true;
        }
    }
    // if input out of bound, just set output to 0
    if (flag_oob) {
        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            outputs[ch] = 0; 
        }
        if (dy_dx) {
            dy_dx += b * D * L * C + level * D * C; // B L D C
            #pragma unroll
            for (uint32_t d = 0; d < D; d++) {
                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    dy_dx[d * C + ch] = 0; 
                }       
            }
        }
        return;
    }

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const uint32_t resolution = (uint32_t)ceil(exp2f(level * S) * H);
    
    // calculate coordinate (always use float for precision!)
    float pos[D];
    float pos_deriv[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        
        // align_corners
        if (align_corners) {
            pos[d] = inputs[d] * (float)(resolution - 1); // [0, resolution - 1]
            pos_grid[d] = min((uint32_t)floorf(pos[d]), resolution - 2); // left-top corner, [0, resolution - 2]
        } else {
            pos[d] = fminf(fmaxf(inputs[d] * (float)resolution - 0.5f, 0.0f), (float)(resolution - 1)); // [-0.5, resolution-0.5] --> [0, resolution - 1]
            pos_grid[d] = (uint32_t)floorf(pos[d]); // left-top corner, [0, resolution - 1]
        }
        pos[d] -= (float)pos_grid[d];

        // smoothstep instead of linear
        if (interp == 1) {
            pos_deriv[d] = smoothstep_derivative(pos[d]);
            pos[d] = smoothstep(pos[d]);
        } else {
            pos_deriv[d] = 1.0f;
        }
    }

    // verification of alignment
    // if (level == L - 1 && b < 4) {
    //     printf("[b=%d, l=%d] pos=(%f, %f)+(%d, %d)\n", b, level, pos[0], pos[1], pos_grid[0], pos_grid[1]);
    // }

    // interpolate
    scalar_t results[C] = {0}; // temp results in register

    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = min(pos_grid[d] + 1, resolution - 1);
            }
        }

        uint32_t index = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid_local);

        // writing to register (fast)
        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            results[ch] += w * grid[index + ch];
        }

        //printf("[b=%d, l=%d] int %d, idx %d, w %f, val %f\n", b, level, idx, index, w, grid[index]);
    }    

    // writing to global memory (slow)
    #pragma unroll
    for (uint32_t ch = 0; ch < C; ch++) {
        outputs[ch] = results[ch]; 
    }

    // prepare dy_dx
    // differentiable (soft) indexing: https://discuss.pytorch.org/t/differentiable-indexing/17647/9
    if (dy_dx) {

        dy_dx += b * D * L * C + level * D * C; // B L D C

        #pragma unroll
        for (uint32_t gd = 0; gd < D; gd++) {

            scalar_t results_grad[C] = {0};

            #pragma unroll
            for (uint32_t idx = 0; idx < (1 << (D - 1)); idx++) {
                float w = (float)(align_corners ? resolution - 1 : resolution);
                uint32_t pos_grid_local[D];

                #pragma unroll
                for (uint32_t nd = 0; nd < D - 1; nd++) {
                    const uint32_t d = (nd >= gd) ? (nd + 1) : nd;

                    if ((idx & (1 << nd)) == 0) {
                        w *= 1 - pos[d];
                        pos_grid_local[d] = pos_grid[d];
                    } else {
                        w *= pos[d];
                        pos_grid_local[d] = min(pos_grid[d] + 1, resolution - 1);
                    }
                }

                pos_grid_local[gd] = pos_grid[gd];
                uint32_t index_left = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid_local);
                pos_grid_local[gd] = min(pos_grid[gd] + 1, resolution - 1);
                uint32_t index_right = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid_local);

                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    results_grad[ch] += w * (grid[index_right + ch] - grid[index_left + ch]) * pos_deriv[gd];
                }
            }

            #pragma unroll
            for (uint32_t ch = 0; ch < C; ch++) {
                dy_dx[gd * C + ch] = results_grad[ch];
            }
        }
    }
}


template <typename scalar_t, uint32_t D, uint32_t C, uint32_t N_C>
__global__ void kernel_grid_backward(
    const scalar_t * __restrict__ grad,
    const float * __restrict__ inputs, 
    const scalar_t * __restrict__ grid, 
    const int * __restrict__ offsets, 
    scalar_t * __restrict__ grad_grid, 
    const uint32_t B, const uint32_t L, const float S, const uint32_t H,
    const uint32_t gridtype,
    const bool align_corners,
    const uint32_t interp
) {
    const uint32_t b = (blockIdx.x * blockDim.x + threadIdx.x) * N_C / C;
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    const uint32_t ch = (blockIdx.x * blockDim.x + threadIdx.x) * N_C - b * C;

    // locate
    grad_grid += offsets[level] * C;
    inputs += b * D;
    grad += level * B * C + b * C + ch; // L, B, C

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const uint32_t resolution = (uint32_t)ceil(exp2f(level * S) * H);

    // check input range (should be in [0, 1])
    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        if (inputs[d] < 0 || inputs[d] > 1) {
            return; // grad is init as 0, so we simply return.
        }
    }

    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        // align_corners
        if (align_corners) {
            pos[d] = inputs[d] * (float)(resolution - 1); // [0, resolution - 1]
            pos_grid[d] = min((uint32_t)floorf(pos[d]), resolution - 2); // left-top corner, [0, resolution - 2]
        } else {
            pos[d] = fminf(fmaxf(inputs[d] * (float)resolution - 0.5f, 0.0f), (float)(resolution - 1)); // [-0.5, resolution-0.5] --> [0, resolution - 1]
            pos_grid[d] = (uint32_t)floorf(pos[d]); // left-top corner, [0, resolution - 1]
        }
        pos[d] -= (float)pos_grid[d];
        // smoothstep instead of linear
        if (interp == 1) {
            pos[d] = smoothstep(pos[d]);
        }
    }

    scalar_t grad_cur[N_C] = {0}; // fetch to register
    #pragma unroll
    for (uint32_t c = 0; c < N_C; c++) {
        grad_cur[c] = grad[c];
    }

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = min(pos_grid[d] + 1, resolution - 1);
            }
        }

        uint32_t index = get_grid_index<D, C>(gridtype, ch, hashmap_size, resolution, pos_grid_local);

        // atomicAdd for __half is slow (especially for large values), so we use __half2 if N_C % 2 == 0
        // TODO: use float which is better than __half, if N_C % 2 != 0
        if (std::is_same<scalar_t, at::Half>::value && N_C % 2 == 0) {
            #pragma unroll
            for (uint32_t c = 0; c < N_C; c += 2) {
                // process two __half at once (by interpreting as a __half2)
                __half2 v = {(__half)(w * grad_cur[c]), (__half)(w * grad_cur[c + 1])};
                atomicAdd((__half2*)&grad_grid[index + c], v);
            }
        // float, or __half when N_C % 2 != 0 (which means C == 1)
        } else {
            #pragma unroll
            for (uint32_t c = 0; c < N_C; c++) {
                atomicAdd(&grad_grid[index + c], w * grad_cur[c]);
            }
        }
    }    
}


template <typename scalar_t, uint32_t D, uint32_t C>
__global__ void kernel_input_backward(
    const scalar_t * __restrict__ grad,
    const scalar_t * __restrict__ dy_dx,  
    scalar_t * __restrict__ grad_inputs, 
    uint32_t B, uint32_t L
) {
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= B * D) return;

    const uint32_t b = t / D;
    const uint32_t d = t - b * D;

    dy_dx += b * L * D * C;

    scalar_t result = 0;
    
    # pragma unroll
    for (int l = 0; l < L; l++) {
        # pragma unroll
        for (int ch = 0; ch < C; ch++) {
            result += grad[l * B * C + b * C + ch] * dy_dx[l * D * C + d * C + ch];
        }
    }

    grad_inputs[t] = result;
}


template <typename scalar_t, uint32_t D>
void kernel_grid_wrapper(const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *outputs, const uint32_t B, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, scalar_t *dy_dx, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    static constexpr uint32_t N_THREAD = 512;
    const dim3 blocks_hashgrid = { div_round_up(B, N_THREAD), max_level, 1 };
    switch (C) {
        case 1: kernel_grid<scalar_t, D, 1><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 2: kernel_grid<scalar_t, D, 2><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 4: kernel_grid<scalar_t, D, 4><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 8: kernel_grid<scalar_t, D, 8><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 16: kernel_grid<scalar_t, D, 16><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 32: kernel_grid<scalar_t, D, 32><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, B, L, S, H, dy_dx, gridtype, align_corners, interp); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, 8, 16 or 32."};
    }
}

// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// outputs: [L, B, C], float (L first, so only one level of hashmap needs to fit into cache at a time.)
// H: base resolution
// dy_dx: [B, L * D * C]
template <typename scalar_t>
void grid_encode_forward_cuda(const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *outputs, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, scalar_t *dy_dx, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    switch (D) {
        case 2: kernel_grid_wrapper<scalar_t, 2>(inputs, embeddings, offsets, outputs, B, C, L, max_level, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 3: kernel_grid_wrapper<scalar_t, 3>(inputs, embeddings, offsets, outputs, B, C, L, max_level, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 4: kernel_grid_wrapper<scalar_t, 4>(inputs, embeddings, offsets, outputs, B, C, L, max_level, S, H, dy_dx, gridtype, align_corners, interp); break;
        case 5: kernel_grid_wrapper<scalar_t, 5>(inputs, embeddings, offsets, outputs, B, C, L, max_level, S, H, dy_dx, gridtype, align_corners, interp); break;
        default: throw std::runtime_error{"GridEncoding: D must be 2, 3, 4 or 5."};
    }   
}

template <typename scalar_t, uint32_t D>
void kernel_grid_backward_wrapper(const scalar_t *grad, const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *grad_embeddings, const uint32_t B, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, scalar_t *dy_dx, scalar_t *grad_inputs, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    static constexpr uint32_t N_THREAD = 256;
    const uint32_t N_C = std::min(2u, C); // n_features_per_thread
    const dim3 blocks_hashgrid = { div_round_up(B * C / N_C, N_THREAD), max_level, 1 };
    switch (C) {
        case 1: 
            kernel_grid_backward<scalar_t, D, 1, 1><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp); 
            if (dy_dx) kernel_input_backward<scalar_t, D, 1><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 2: 
            kernel_grid_backward<scalar_t, D, 2, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp);
            if (dy_dx) kernel_input_backward<scalar_t, D, 2><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 4: 
            kernel_grid_backward<scalar_t, D, 4, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp);
            if (dy_dx) kernel_input_backward<scalar_t, D, 4><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 8: 
            kernel_grid_backward<scalar_t, D, 8, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp);
            if (dy_dx) kernel_input_backward<scalar_t, D, 8><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 16: 
            kernel_grid_backward<scalar_t, D, 16, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp);
            if (dy_dx) kernel_input_backward<scalar_t, D, 16><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        case 32: 
            kernel_grid_backward<scalar_t, D, 32, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, B, L, S, H, gridtype, align_corners, interp);
            if (dy_dx) kernel_input_backward<scalar_t, D, 32><<<div_round_up(B * D, N_THREAD), N_THREAD>>>(grad, dy_dx, grad_inputs, B, L);
            break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, 8, 16 or 32."};
    }
}


// grad: [L, B, C], float
// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// grad_embeddings: [sO, C]
// H: base resolution
template <typename scalar_t>
void grid_encode_backward_cuda(const scalar_t *grad, const float *inputs, const scalar_t *embeddings, const int *offsets, scalar_t *grad_embeddings, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, scalar_t *dy_dx, scalar_t *grad_inputs, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    switch (D) {
        case 2: kernel_grid_backward_wrapper<scalar_t, 2>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, max_level, S, H, dy_dx, grad_inputs, gridtype, align_corners, interp); break;
        case 3: kernel_grid_backward_wrapper<scalar_t, 3>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, max_level, S, H, dy_dx, grad_inputs, gridtype, align_corners, interp); break;
        case 4: kernel_grid_backward_wrapper<scalar_t, 4>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, max_level, S, H, dy_dx, grad_inputs, gridtype, align_corners, interp); break;
        case 5: kernel_grid_backward_wrapper<scalar_t, 5>(grad, inputs, embeddings, offsets, grad_embeddings, B, C, L, max_level, S, H, dy_dx, grad_inputs, gridtype, align_corners, interp); break;
        default: throw std::runtime_error{"GridEncoding: D must be 2, 3, 4 or 5."};
    }
}



void grid_encode_forward(const at::Tensor inputs, const at::Tensor embeddings, const at::Tensor offsets, at::Tensor outputs, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, at::optional<at::Tensor> dy_dx, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    CHECK_CUDA(inputs);
    CHECK_CUDA(embeddings);
    CHECK_CUDA(offsets);
    CHECK_CUDA(outputs);
    // CHECK_CUDA(dy_dx);
    
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(embeddings);
    CHECK_CONTIGUOUS(offsets);
    CHECK_CONTIGUOUS(outputs);
    // CHECK_CONTIGUOUS(dy_dx);

    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(embeddings);
    CHECK_IS_INT(offsets);
    CHECK_IS_FLOATING(outputs);
    // CHECK_IS_FLOATING(dy_dx);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    embeddings.scalar_type(), "grid_encode_forward", ([&] {
        grid_encode_forward_cuda<scalar_t>(inputs.data_ptr<float>(), embeddings.data_ptr<scalar_t>(), offsets.data_ptr<int>(), outputs.data_ptr<scalar_t>(), B, D, C, L, max_level, S, H, dy_dx.has_value() ? dy_dx.value().data_ptr<scalar_t>() : nullptr, gridtype, align_corners, interp);
    }));
}

void grid_encode_backward(const at::Tensor grad, const at::Tensor inputs, const at::Tensor embeddings, const at::Tensor offsets, at::Tensor grad_embeddings, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t max_level, const float S, const uint32_t H, const at::optional<at::Tensor> dy_dx, at::optional<at::Tensor> grad_inputs, const uint32_t gridtype, const bool align_corners, const uint32_t interp) {
    CHECK_CUDA(grad);
    CHECK_CUDA(inputs);
    CHECK_CUDA(embeddings);
    CHECK_CUDA(offsets);
    CHECK_CUDA(grad_embeddings);
    // CHECK_CUDA(dy_dx);
    // CHECK_CUDA(grad_inputs);
    
    CHECK_CONTIGUOUS(grad);
    CHECK_CONTIGUOUS(inputs);
    CHECK_CONTIGUOUS(embeddings);
    CHECK_CONTIGUOUS(offsets);
    CHECK_CONTIGUOUS(grad_embeddings);
    // CHECK_CONTIGUOUS(dy_dx);
    // CHECK_CONTIGUOUS(grad_inputs);

    CHECK_IS_FLOATING(grad);
    CHECK_IS_FLOATING(inputs);
    CHECK_IS_FLOATING(embeddings);
    CHECK_IS_INT(offsets);
    CHECK_IS_FLOATING(grad_embeddings);
    // CHECK_IS_FLOATING(dy_dx);
    // CHECK_IS_FLOATING(grad_inputs);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad.scalar_type(), "grid_encode_backward", ([&] {
        grid_encode_backward_cuda<scalar_t>(grad.data_ptr<scalar_t>(), inputs.data_ptr<float>(), embeddings.data_ptr<scalar_t>(), offsets.data_ptr<int>(), grad_embeddings.data_ptr<scalar_t>(), B, D, C, L, max_level, S, H, dy_dx.has_value() ? dy_dx.value().data_ptr<scalar_t>() : nullptr, grad_inputs.has_value() ? grad_inputs.value().data_ptr<scalar_t>() : nullptr, gridtype, align_corners, interp);
    }));
    
}


template <typename scalar_t, uint32_t D, uint32_t C>
__global__ void kernel_grad_tv(
    const scalar_t * __restrict__ inputs,
    const scalar_t * __restrict__ grid, 
    scalar_t * __restrict__ grad, 
    const int * __restrict__ offsets, 
    const float weight,
    const uint32_t B, const uint32_t L, const float S, const uint32_t H,
    const uint32_t gridtype,
    const bool align_corners
) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= B) return;

    const uint32_t level = blockIdx.y;
    
    // locate
    inputs += b * D;
    grid += (uint32_t)offsets[level] * C;
    grad += (uint32_t)offsets[level] * C;

    // check input range (should be in [0, 1])
    bool flag_oob = false;
    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        if (inputs[d] < 0 || inputs[d] > 1) {
            flag_oob = true;
        }
    }

    // if input out of bound, do nothing
    if (flag_oob) return;

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    const uint32_t resolution = (uint32_t)ceil(exp2f(level * S) * H);
    
    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D]; // [0, resolution]

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        // align_corners
        if (align_corners) {
            pos[d] = inputs[d] * (float)(resolution - 1); // [0, resolution - 1]
            pos_grid[d] = min((uint32_t)floorf(pos[d]), resolution - 2); // left-top corner, [0, resolution - 2]
        } else {
            pos[d] = fminf(fmaxf(inputs[d] * (float)resolution - 0.5f, 0.0f), (float)(resolution - 1)); // [-0.5, resolution-0.5] --> [0, resolution - 1]
            pos_grid[d] = (uint32_t)floorf(pos[d]); // left-top corner, [0, resolution - 1]
        }
    }

    //printf("[b=%d, l=%d] pos=(%f, %f)+(%d, %d)\n", b, level, pos[0], pos[1], pos_grid[0], pos_grid[1]);

    // total variation on pos_grid
    scalar_t results[C] = {0}; // temp results in register
    scalar_t idelta[C] = {0};

    uint32_t index = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid);

    scalar_t w = weight / (2 * D);

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {

        uint32_t cur_d = pos_grid[d];
        scalar_t grad_val;

        // right side
        if (cur_d < resolution) {
            pos_grid[d] = cur_d + 1;
            uint32_t index_right = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid);

            #pragma unroll
            for (uint32_t ch = 0; ch < C; ch++) {
                grad_val = (grid[index + ch] - grid[index_right + ch]);
                results[ch] += grad_val;
                idelta[ch] += grad_val * grad_val;
            }
        }

        // left side
        if (cur_d > 0) {
            pos_grid[d] = cur_d - 1;
            uint32_t index_left = get_grid_index<D, C>(gridtype, 0, hashmap_size, resolution, pos_grid);

            #pragma unroll
            for (uint32_t ch = 0; ch < C; ch++) {
                grad_val = (grid[index + ch] - grid[index_left + ch]);
                results[ch] += grad_val;
                idelta[ch] += grad_val * grad_val;
            }
        }

        // reset
        pos_grid[d] = cur_d;
    }

    // writing to global memory (slow)
    #pragma unroll
    for (uint32_t ch = 0; ch < C; ch++) {
        // index may collide, so use atomic!
        atomicAdd(&grad[index + ch], w * results[ch] * rsqrtf(idelta[ch] + 1e-9f));
    }

}


template <typename scalar_t, uint32_t D>
void kernel_grad_tv_wrapper(const scalar_t *inputs, const scalar_t *embeddings, scalar_t *grad, const int *offsets, const float weight, const uint32_t B, const uint32_t C, const uint32_t L, const float S, const uint32_t H, const uint32_t gridtype, const bool align_corners) {
    static constexpr uint32_t N_THREAD = 512;
    const dim3 blocks_hashgrid = { div_round_up(B, N_THREAD), L, 1 };
    switch (C) {
        case 1: kernel_grad_tv<scalar_t, D, 1><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        case 2: kernel_grad_tv<scalar_t, D, 2><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        case 4: kernel_grad_tv<scalar_t, D, 4><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        case 8: kernel_grad_tv<scalar_t, D, 8><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        case 16: kernel_grad_tv<scalar_t, D, 16><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        case 32: kernel_grad_tv<scalar_t, D, 32><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, grad, offsets, weight, B, L, S, H, gridtype, align_corners); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, 8, 16 or 32."};
    }
}


template <typename scalar_t>
void grad_total_variation_cuda(const scalar_t *inputs, const scalar_t *embeddings, scalar_t *grad, const int *offsets, const float weight, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, const uint32_t gridtype, const bool align_corners) {
    switch (D) {
        case 2: kernel_grad_tv_wrapper<scalar_t, 2>(inputs, embeddings, grad, offsets, weight, B, C, L, S, H, gridtype, align_corners); break;
        case 3: kernel_grad_tv_wrapper<scalar_t, 3>(inputs, embeddings, grad, offsets, weight, B, C, L, S, H, gridtype, align_corners); break;
        case 4: kernel_grad_tv_wrapper<scalar_t, 4>(inputs, embeddings, grad, offsets, weight, B, C, L, S, H, gridtype, align_corners); break;
        case 5: kernel_grad_tv_wrapper<scalar_t, 5>(inputs, embeddings, grad, offsets, weight, B, C, L, S, H, gridtype, align_corners); break;
        default: throw std::runtime_error{"GridEncoding: D must be 2, 3, 4, or 5."};
    }   
}


void grad_total_variation(const at::Tensor inputs, const at::Tensor embeddings, at::Tensor grad, const at::Tensor offsets, const float weight, const uint32_t B, const uint32_t D, const uint32_t C, const uint32_t L, const float S, const uint32_t H, const uint32_t gridtype, const bool align_corners) {

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    embeddings.scalar_type(), "grad_total_variation", ([&] {
        grad_total_variation_cuda<scalar_t>(inputs.data_ptr<scalar_t>(), embeddings.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), offsets.data_ptr<int>(), weight, B, D, C, L, S, H, gridtype, align_corners);
    }));
}

template <typename scalar_t>
__global__ void kernel_grad_wd(
    const scalar_t * __restrict__ grid, 
    scalar_t * __restrict__ grad, 
    const int * __restrict__ offsets, 
    const float weight,
    const uint32_t B, const uint32_t L, const uint32_t C
) {
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= B * C) return;

    // locate
    grid += b;
    grad += b;

    // decide in which level is this thread... 
    uint32_t level = 0;
    const uint32_t n = b / C;
    // binary search b in offsets
    uint32_t l = 0, r = L;
    while (l < r) {
        uint32_t m = (l + r) / 2;
        if (offsets[m] <= n) {
            level = m;
            l = m + 1;
        } else {
            r = m;
        }
    }

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    grad[0] += 2 * weight * grid[0] / hashmap_size;
}

void grad_weight_decay(const at::Tensor embeddings, at::Tensor grad, const at::Tensor offsets, const float weight, const uint32_t B, const uint32_t C, const uint32_t L) {

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    embeddings.scalar_type(), "grad_weight_decay", ([&] {
        static constexpr uint32_t N_THREAD = 1024;
        const dim3 blocks_hashgrid = { div_round_up(B * C, N_THREAD), 1, 1 };
        kernel_grad_wd<scalar_t><<<blocks_hashgrid, N_THREAD>>>(embeddings.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), offsets.data_ptr<int>(), weight, B, L, C);
    }));
}