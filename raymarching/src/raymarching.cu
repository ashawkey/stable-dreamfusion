#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <cstdio>
#include <stdint.h>
#include <stdexcept>
#include <limits>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be a contiguous tensor")
#define CHECK_IS_INT(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Int, #x " must be an int tensor")
#define CHECK_IS_FLOATING(x) TORCH_CHECK(x.scalar_type() == at::ScalarType::Float || x.scalar_type() == at::ScalarType::Half || x.scalar_type() == at::ScalarType::Double, #x " must be a floating tensor")


inline constexpr __device__ float SQRT3() { return 1.7320508075688772f; }
inline constexpr __device__ float RSQRT3() { return 0.5773502691896258f; }
inline constexpr __device__ float PI() { return 3.141592653589793f; }
inline constexpr __device__ float RPI() { return 0.3183098861837907f; }


template <typename T>
inline __host__ __device__ T div_round_up(T val, T divisor) {
    return (val + divisor - 1) / divisor;
}

inline __host__ __device__ float signf(const float x) {
    return copysignf(1.0, x);
}

inline __host__ __device__ float clamp(const float x, const float min, const float max) {
    return fminf(max, fmaxf(min, x));
}

inline __host__ __device__ void swapf(float& a, float& b) {
    float c = a; a = b; b = c;
}

inline __device__ int mip_from_pos(const float x, const float y, const float z, const float max_cascade) {
    const float mx = fmaxf(fabsf(x), fmaxf(fabsf(y), fabsf(z)));
    int exponent;
    frexpf(mx, &exponent); // [0, 0.5) --> -1, [0.5, 1) --> 0, [1, 2) --> 1, [2, 4) --> 2, ...
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __device__ int mip_from_dt(const float dt, const float H, const float max_cascade) {
    const float mx = dt * H * 0.5;
    int exponent;
    frexpf(mx, &exponent);
    return fminf(max_cascade - 1, fmaxf(0, exponent));
}

inline __host__ __device__ uint32_t __expand_bits(uint32_t v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

inline __host__ __device__ uint32_t __morton3D(uint32_t x, uint32_t y, uint32_t z)
{
	uint32_t xx = __expand_bits(x);
	uint32_t yy = __expand_bits(y);
	uint32_t zz = __expand_bits(z);
	return xx | (yy << 1) | (zz << 2);
}

inline __host__ __device__ uint32_t __morton3D_invert(uint32_t x)
{
	x = x & 0x49249249;
	x = (x | (x >> 2)) & 0xc30c30c3;
	x = (x | (x >> 4)) & 0x0f00f00f;
	x = (x | (x >> 8)) & 0xff0000ff;
	x = (x | (x >> 16)) & 0x0000ffff;
	return x;
}


////////////////////////////////////////////////////
/////////////           utils          /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// nears/fars: [N]
// scalar_t should always be float in use.
template <typename scalar_t>
__global__ void kernel_near_far_from_aabb(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const scalar_t * __restrict__ aabb,
    const uint32_t N,
    const float min_near,
    scalar_t * nears, scalar_t * fars
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // get near far (assume cube scene)
    float near = (aabb[0] - ox) * rdx;
    float far = (aabb[3] - ox) * rdx;
    if (near > far) swapf(near, far);

    float near_y = (aabb[1] - oy) * rdy;
    float far_y = (aabb[4] - oy) * rdy;
    if (near_y > far_y) swapf(near_y, far_y);

    if (near > far_y || near_y > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_y > near) near = near_y;
    if (far_y < far) far = far_y;

    float near_z = (aabb[2] - oz) * rdz;
    float far_z = (aabb[5] - oz) * rdz;
    if (near_z > far_z) swapf(near_z, far_z);

    if (near > far_z || near_z > far) {
        nears[n] = fars[n] = std::numeric_limits<scalar_t>::max();
        return;
    }

    if (near_z > near) near = near_z;
    if (far_z < far) far = far_z;

    if (near < min_near) near = min_near;

    nears[n] = near;
    fars[n] = far;
}


void near_far_from_aabb(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor aabb, const uint32_t N, const float min_near, at::Tensor nears, at::Tensor fars) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "near_far_from_aabb", ([&] {
        kernel_near_far_from_aabb<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), aabb.data_ptr<scalar_t>(), N, min_near, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>());
    }));
}


// rays_o/d: [N, 3]
// radius: float
// coords: [N, 2]
template <typename scalar_t>
__global__ void kernel_sph_from_ray(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,
    const float radius,
    const uint32_t N,
    scalar_t * coords
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    coords += n * 2;

    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    // const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;

    // solve t from || o + td || = radius
    const float A = dx * dx + dy * dy + dz * dz;
    const float B = ox * dx + oy * dy + oz * dz; // in fact B / 2
    const float C = ox * ox + oy * oy + oz * oz - radius * radius;

    const float t = (- B + sqrtf(B * B - A * C)) / A; // always use the larger solution (positive)

    // solve theta, phi (assume y is the up axis)
    const float x = ox + t * dx, y = oy + t * dy, z = oz + t * dz;
    const float theta = atan2(sqrtf(x * x + z * z), y); // [0, PI)
    const float phi = atan2(z, x); // [-PI, PI)

    // normalize to [-1, 1]
    coords[0] = 2 * theta * RPI() - 1;
    coords[1] = phi * RPI();
}


void sph_from_ray(const at::Tensor rays_o, const at::Tensor rays_d, const float radius, const uint32_t N, at::Tensor coords) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "sph_from_ray", ([&] {
        kernel_sph_from_ray<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), radius, N, coords.data_ptr<scalar_t>());
    }));
}


// coords: int32, [N, 3]
// indices: int32, [N]
__global__ void kernel_morton3D(
    const int * __restrict__ coords,
    const uint32_t N,
    int * indices
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;
    indices[n] = __morton3D(coords[0], coords[1], coords[2]);
}


void morton3D(const at::Tensor coords, const uint32_t N, at::Tensor indices) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D<<<div_round_up(N, N_THREAD), N_THREAD>>>(coords.data_ptr<int>(), N, indices.data_ptr<int>());
}


// indices: int32, [N]
// coords: int32, [N, 3]
__global__ void kernel_morton3D_invert(
    const int * __restrict__ indices,
    const uint32_t N,
    int * coords
) {
    // parallel
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    coords += n * 3;

    const int ind = indices[n];

    coords[0] = __morton3D_invert(ind >> 0);
    coords[1] = __morton3D_invert(ind >> 1);
    coords[2] = __morton3D_invert(ind >> 2);
}


void morton3D_invert(const at::Tensor indices, const uint32_t N, at::Tensor coords) {
    static constexpr uint32_t N_THREAD = 128;
    kernel_morton3D_invert<<<div_round_up(N, N_THREAD), N_THREAD>>>(indices.data_ptr<int>(), N, coords.data_ptr<int>());
}


// grid: float, [C, H, H, H]
// N: int, C * H * H * H / 8
// density_thresh: float
// bitfield: uint8, [N]
template <typename scalar_t>
__global__ void kernel_packbits(
    const scalar_t * __restrict__ grid,
    const uint32_t N,
    const float density_thresh,
    uint8_t * bitfield
) {
    // parallel per byte
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    grid += n * 8;

    uint8_t bits = 0;

    #pragma unroll
    for (uint8_t i = 0; i < 8; i++) {
        bits |= (grid[i] > density_thresh) ? ((uint8_t)1 << i) : 0;
    }

    bitfield[n] = bits;
}


void packbits(const at::Tensor grid, const uint32_t N, const float density_thresh, at::Tensor bitfield) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grid.scalar_type(), "packbits", ([&] {
        kernel_packbits<<<div_round_up(N, N_THREAD), N_THREAD>>>(grid.data_ptr<scalar_t>(), N, density_thresh, bitfield.data_ptr<uint8_t>());
    }));
}


__global__ void kernel_flatten_rays(
    const int * __restrict__ rays,
    const uint32_t N, const uint32_t M,
    int * res
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate
    uint32_t offset = rays[n * 2];
    uint32_t num_steps = rays[n * 2 + 1];

    // write to res
    res += offset;
    for (int i = 0; i < num_steps; i++) res[i] = n;
}

void flatten_rays(const at::Tensor rays, const uint32_t N, const uint32_t M, at::Tensor res) {

    static constexpr uint32_t N_THREAD = 128;

    kernel_flatten_rays<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays.data_ptr<int>(), N, M, res.data_ptr<int>());
}

////////////////////////////////////////////////////
/////////////         training         /////////////
////////////////////////////////////////////////////

// rays_o/d: [N, 3]
// grid: [CHHH / 8]
// xyzs, dirs, ts: [M, 3], [M, 3], [M, 2]
// dirs: [M, 3]
// rays: [N, 3], idx, offset, num_steps
template <typename scalar_t>
__global__ void kernel_march_rays_train(
    const scalar_t * __restrict__ rays_o,
    const scalar_t * __restrict__ rays_d,  
    const uint8_t * __restrict__ grid,
    const float bound, const bool contract,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t N, const uint32_t C, const uint32_t H,
    const scalar_t* __restrict__ nears, 
    const scalar_t* __restrict__ fars,
    scalar_t * xyzs, scalar_t * dirs, scalar_t * ts,
    int * rays,
    int * counter,
    const scalar_t* __restrict__ noises
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // is first pass running.
    const bool first_pass = (xyzs == nullptr);

    // locate
    rays_o += n * 3;
    rays_d += n * 3;
    rays += n * 2;

    uint32_t num_steps = max_steps;

    if (!first_pass) {
        uint32_t point_index = rays[0];
        num_steps = rays[1];
        xyzs += point_index * 3;
        dirs += point_index * 3;
        ts += point_index * 2;
    }

    // ray marching
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;

    const float near = nears[n];
    const float far = fars[n];
    const float noise = noises[n];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * bound / H;
    // const float dt_max = 1e10f;
    
    float t0 = near;
    t0 += clamp(t0 * dt_gamma, dt_min, dt_max) * noise;
    float t = t0;
    uint32_t step = 0;

    //if (t < far) printf("valid ray %d t=%f near=%f far=%f \n", n, t, near, far);
    
    while (t < far && step < num_steps) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1.0f, level), bound);
        const float mip_rbound = 1 / mip_bound;

        // contraction
        float cx = x, cy = y, cz = z;
        const float mag = fmaxf(fabsf(x), fmaxf(fabsf(y), fabsf(z)));
        if (contract && mag > 1) {
            // L-INF norm
            const float Linf_scale = (2 - 1 / mag) / mag;
            cx *= Linf_scale;
            cy *= Linf_scale;
            cz *= Linf_scale;
        }
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (cx * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (cy * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (cz * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        //if (n == 0) printf("t=%f density=%f vs thresh=%f step=%d\n", t, density, density_thresh, step);

        if (occ) {
            step++;
            t += dt;
            if (!first_pass) {
                xyzs[0] = cx; // write contracted coordinates!
                xyzs[1] = cy;
                xyzs[2] = cz;
                dirs[0] = dx;
                dirs[1] = dy;
                dirs[2] = dz;
                ts[0] = t;
                ts[1] = dt;
                xyzs += 3;
                dirs += 3;
                ts += 2;
            }
        // contraction case: cannot apply voxel skipping.
        } else if (contract && mag > 1) {
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - cx) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - cy) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - cz) * rdz;

            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                dt = clamp(t * dt_gamma, dt_min, dt_max);
                t += dt;
            } while (t < tt);
        }
    }

    //printf("[n=%d] step=%d, near=%f, far=%f, dt=%f, num_steps=%f\n", n, step, near, far, dt_min, (far - near) / dt_min);

    // write rays
    if (first_pass) {
        uint32_t point_index = atomicAdd(counter, step);
        rays[0] = point_index;
        rays[1] = step;
    }
}

void march_rays_train(const at::Tensor rays_o, const at::Tensor rays_d, const at::Tensor grid, const float bound, const bool contract, const float dt_gamma, const uint32_t max_steps, const uint32_t N, const uint32_t C, const uint32_t H, const at::Tensor nears, const at::Tensor fars, at::optional<at::Tensor> xyzs, at::optional<at::Tensor> dirs, at::optional<at::Tensor> ts, at::Tensor rays, at::Tensor counter, at::Tensor noises) {

    static constexpr uint32_t N_THREAD = 128;
    
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays_train", ([&] {
        kernel_march_rays_train<<<div_round_up(N, N_THREAD), N_THREAD>>>(rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), grid.data_ptr<uint8_t>(), bound, contract, dt_gamma, max_steps, N, C, H, nears.data_ptr<scalar_t>(), fars.data_ptr<scalar_t>(),
            xyzs.has_value() ? xyzs.value().data_ptr<scalar_t>() : nullptr,
            dirs.has_value() ? dirs.value().data_ptr<scalar_t>() : nullptr,
            ts.has_value() ? ts.value().data_ptr<scalar_t>() : nullptr,
            rays.data_ptr<int>(), counter.data_ptr<int>(), noises.data_ptr<scalar_t>());
    }));
}


// sigmas: [M]
// rgbs: [M, 3]
// ts: [M, 2]
// rays: [N, 2], offset, num_steps
// weights: [M]
// weights_sum: [N], final pixel alpha
// depth: [N,]
// image: [N, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_forward(
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs,  
    const scalar_t * __restrict__ ts,
    const int * __restrict__ rays,
    const uint32_t M, const uint32_t N, const float T_thresh, const bool binarize,
    scalar_t * weights,
    scalar_t * weights_sum,
    scalar_t * depth,
    scalar_t * image
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t offset = rays[n * 2];
    uint32_t num_steps = rays[n * 2 + 1];

    // empty ray, or ray that exceed max step count.
    if (num_steps == 0 || offset + num_steps > M) {
        weights_sum[n] = 0;
        depth[n] = 0;
        image[n * 3] = 0;
        image[n * 3 + 1] = 0;
        image[n * 3 + 2] = 0;
        return;
    }

    ts += offset * 2;
    weights += offset;
    sigmas += offset;
    rgbs += offset * 3;

    // accumulate 
    uint32_t step = 0;

    float T = 1.0f;
    float r = 0, g = 0, b = 0, ws = 0, d = 0;

    while (step < num_steps) {

        const float real_alpha = 1.0f - __expf(- sigmas[0] * ts[1]);
        const float alpha = binarize ? (real_alpha > 0.5 ? 1.0 : 0.0) : real_alpha;
        const float weight = alpha * T;

        weights[0] = weight;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;
        d += weight * ts[0];
        
        T *= 1.0f - alpha;

        // minimal remained transmittence
        if (T < T_thresh) break;

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // locate
        weights++;
        sigmas++;
        rgbs += 3;
        ts += 2;

        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // write
    weights_sum[n] = ws; // weights_sum
    depth[n] = d;
    image[n * 3] = r;
    image[n * 3 + 1] = g;
    image[n * 3 + 2] = b;
}


void composite_rays_train_forward(const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ts, const at::Tensor rays, const uint32_t M, const uint32_t N, const float T_thresh, const bool binarize, at::Tensor weights, at::Tensor weights_sum, at::Tensor depth, at::Tensor image) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    sigmas.scalar_type(), "composite_rays_train_forward", ([&] {
        kernel_composite_rays_train_forward<<<div_round_up(N, N_THREAD), N_THREAD>>>(sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ts.data_ptr<scalar_t>(), rays.data_ptr<int>(), M, N, T_thresh, binarize, weights.data_ptr<scalar_t>(), weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}


// grad_weights: [M,]
// grad_weights_sum: [N,]
// grad_image: [N, 3]
// grad_depth: [N,]
// sigmas: [M]
// rgbs: [M, 3]
// ts: [M, 2]
// rays: [N, 2], offset, num_steps
// weights_sum: [N,], weights_sum here 
// image: [N, 3]
// grad_sigmas: [M]
// grad_rgbs: [M, 3]
template <typename scalar_t>
__global__ void kernel_composite_rays_train_backward(
    const scalar_t * __restrict__ grad_weights,
    const scalar_t * __restrict__ grad_weights_sum,
    const scalar_t * __restrict__ grad_depth,
    const scalar_t * __restrict__ grad_image,
    const scalar_t * __restrict__ sigmas,
    const scalar_t * __restrict__ rgbs, 
    const scalar_t * __restrict__ ts,
    const int * __restrict__ rays,
    const scalar_t * __restrict__ weights_sum,
    const scalar_t * __restrict__ depth,
    const scalar_t * __restrict__ image,
    const uint32_t M, const uint32_t N, const float T_thresh, const bool binarize,
    scalar_t * grad_sigmas,
    scalar_t * grad_rgbs
) {
    // parallel per ray
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= N) return;

    // locate 
    uint32_t offset = rays[n * 2];
    uint32_t num_steps = rays[n * 2 + 1];

    if (num_steps == 0 || offset + num_steps > M) return;

    grad_weights += offset;
    grad_weights_sum += n;
    grad_depth += n;
    grad_image += n * 3;
    weights_sum += n;
    depth += n;
    image += n * 3;
    sigmas += offset;
    rgbs += offset * 3;
    ts += offset * 2;
    grad_sigmas += offset;
    grad_rgbs += offset * 3;

    // accumulate 
    uint32_t step = 0;
    
    float T = 1.0f;
    const float r_final = image[0], g_final = image[1], b_final = image[2], ws_final = weights_sum[0], d_final = depth[0];
    float r = 0, g = 0, b = 0, ws = 0, d = 0;

    while (step < num_steps) {
        
        const float real_alpha = 1.0f - __expf(- sigmas[0] * ts[1]);
        const float alpha = binarize ? (real_alpha > 0.5 ? 1.0 : 0.0) : real_alpha;
        const float weight = alpha * T;

        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];
        ws += weight;
        d += weight * ts[0];

        T *= 1.0f - alpha;
        
        // check https://note.kiui.moe/others/nerf_gradient/ for the gradient calculation.
        // write grad_rgbs
        grad_rgbs[0] = grad_image[0] * weight;
        grad_rgbs[1] = grad_image[1] * weight;
        grad_rgbs[2] = grad_image[2] * weight;

        // write grad_sigmas
        grad_sigmas[0] = ts[1] * (
            grad_image[0] * (T * rgbs[0] - (r_final - r)) + 
            grad_image[1] * (T * rgbs[1] - (g_final - g)) + 
            grad_image[2] * (T * rgbs[2] - (b_final - b)) +
            (grad_weights_sum[0] + grad_weights[0]) * (T - (ws_final - ws)) + 
            grad_depth[0] * (T * ts[0] - (d_final - d))
        );

        //printf("[n=%d] num_steps=%d, T=%f, grad_sigmas=%f, r_final=%f, r=%f\n", n, step, T, grad_sigmas[0], r_final, r);
        // minimal remained transmittence
        if (T < T_thresh) break;
        
        // locate
        sigmas++;
        rgbs += 3;
        ts += 2;
        grad_weights++;
        grad_sigmas++;
        grad_rgbs += 3;

        step++;
    }
}


void composite_rays_train_backward(const at::Tensor grad_weights, const at::Tensor grad_weights_sum, const at::Tensor grad_depth, const at::Tensor grad_image, const at::Tensor sigmas, const at::Tensor rgbs, const at::Tensor ts, const at::Tensor rays, const at::Tensor weights_sum, const at::Tensor depth, const at::Tensor image, const uint32_t M, const uint32_t N, const float T_thresh, const bool binarize, at::Tensor grad_sigmas, at::Tensor grad_rgbs) {

    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_image.scalar_type(), "composite_rays_train_backward", ([&] {
        kernel_composite_rays_train_backward<<<div_round_up(N, N_THREAD), N_THREAD>>>(grad_weights.data_ptr<scalar_t>(), grad_weights_sum.data_ptr<scalar_t>(), grad_depth.data_ptr<scalar_t>(), grad_image.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ts.data_ptr<scalar_t>(), rays.data_ptr<int>(), weights_sum.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>(), M, N, T_thresh, binarize, grad_sigmas.data_ptr<scalar_t>(), grad_rgbs.data_ptr<scalar_t>());
    }));
}


////////////////////////////////////////////////////
/////////////          infernce        /////////////
////////////////////////////////////////////////////

template <typename scalar_t>
__global__ void kernel_march_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const int* __restrict__ rays_alive, 
    const scalar_t* __restrict__ rays_t, 
    const scalar_t* __restrict__ rays_o, 
    const scalar_t* __restrict__ rays_d, 
    const float bound, const bool contract,
    const float dt_gamma, const uint32_t max_steps,
    const uint32_t C, const uint32_t H,
    const uint8_t * __restrict__ grid,
    const scalar_t* __restrict__ nears,
    const scalar_t* __restrict__ fars,
    scalar_t* xyzs, scalar_t* dirs, scalar_t* ts,
    const scalar_t* __restrict__ noises
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    const float noise = noises[n];
    
    // locate
    rays_o += index * 3;
    rays_d += index * 3;
    xyzs += n * n_step * 3;
    dirs += n * n_step * 3;
    ts += n * n_step * 2;
    
    const float ox = rays_o[0], oy = rays_o[1], oz = rays_o[2];
    const float dx = rays_d[0], dy = rays_d[1], dz = rays_d[2];
    const float rdx = 1 / dx, rdy = 1 / dy, rdz = 1 / dz;
    const float rH = 1 / (float)H;
    const float H3 = H * H * H;
    
    const float near = nears[index], far = fars[index];

    const float dt_min = 2 * SQRT3() / max_steps;
    const float dt_max = 2 * SQRT3() * bound / H;
    // const float dt_max = 1e10f;

    // march for n_step steps, record points
    float t = rays_t[index];
    t += clamp(t * dt_gamma, dt_min, dt_max) * noise;
    uint32_t step = 0;

    while (t < far && step < n_step) {
        // current point
        const float x = clamp(ox + t * dx, -bound, bound);
        const float y = clamp(oy + t * dy, -bound, bound);
        const float z = clamp(oz + t * dz, -bound, bound);

        float dt = clamp(t * dt_gamma, dt_min, dt_max);

        // get mip level
        const int level = max(mip_from_pos(x, y, z, C), mip_from_dt(dt, H, C)); // range in [0, C - 1]

        const float mip_bound = fminf(scalbnf(1, level), bound);
        const float mip_rbound = 1 / mip_bound;
        
        // contraction
        float cx = x, cy = y, cz = z;
        const float mag = fmaxf(fabsf(x), fmaxf(fabsf(y), fabsf(z)));
        if (contract && mag > 1) {
            // L-INF norm
            const float Linf_scale = (2 - 1 / mag) / mag;
            cx *= Linf_scale;
            cy *= Linf_scale;
            cz *= Linf_scale;
        }
        
        // convert to nearest grid position
        const int nx = clamp(0.5 * (cx * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int ny = clamp(0.5 * (cy * mip_rbound + 1) * H, 0.0f, (float)(H - 1));
        const int nz = clamp(0.5 * (cz * mip_rbound + 1) * H, 0.0f, (float)(H - 1));

        const uint32_t index = level * H3 + __morton3D(nx, ny, nz);
        const bool occ = grid[index / 8] & (1 << (index % 8));

        // if occpuied, advance a small step, and write to output
        if (occ) {
            // write step
            xyzs[0] = cx;
            xyzs[1] = cy;
            xyzs[2] = cz;
            dirs[0] = dx;
            dirs[1] = dy;
            dirs[2] = dz;
            // calc dt
            t += dt;
            ts[0] = t;
            ts[1] = dt;
            // step
            xyzs += 3;
            dirs += 3;
            ts += 2;
            step++;

        // contraction case
        } else if (contract && mag > 1) {
            t += dt;
        // else, skip a large step (basically skip a voxel grid)
        } else {
            // calc distance to next voxel
            const float tx = (((nx + 0.5f + 0.5f * signf(dx)) * rH * 2 - 1) * mip_bound - cx) * rdx;
            const float ty = (((ny + 0.5f + 0.5f * signf(dy)) * rH * 2 - 1) * mip_bound - cy) * rdy;
            const float tz = (((nz + 0.5f + 0.5f * signf(dz)) * rH * 2 - 1) * mip_bound - cz) * rdz;
            const float tt = t + fmaxf(0.0f, fminf(tx, fminf(ty, tz)));
            // step until next voxel
            do { 
                dt = clamp(t * dt_gamma, dt_min, dt_max);
                t += dt;
            } while (t < tt);
        }
    }
}


void march_rays(const uint32_t n_alive, const uint32_t n_step, const at::Tensor rays_alive, const at::Tensor rays_t, const at::Tensor rays_o, const at::Tensor rays_d, const float bound, const bool contract, const float dt_gamma, const uint32_t max_steps, const uint32_t C, const uint32_t H, const at::Tensor grid, const at::Tensor near, const at::Tensor far, at::Tensor xyzs, at::Tensor dirs, at::Tensor ts, at::Tensor noises) {
    static constexpr uint32_t N_THREAD = 128;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    rays_o.scalar_type(), "march_rays", ([&] {
        kernel_march_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), rays_o.data_ptr<scalar_t>(), rays_d.data_ptr<scalar_t>(), bound, contract, dt_gamma, max_steps, C, H, grid.data_ptr<uint8_t>(), near.data_ptr<scalar_t>(), far.data_ptr<scalar_t>(), xyzs.data_ptr<scalar_t>(), dirs.data_ptr<scalar_t>(), ts.data_ptr<scalar_t>(), noises.data_ptr<scalar_t>());
    }));
}


template <typename scalar_t>
__global__ void kernel_composite_rays(
    const uint32_t n_alive, 
    const uint32_t n_step, 
    const float T_thresh, const bool binarize,
    int* rays_alive, 
    scalar_t* rays_t, 
    const scalar_t* __restrict__ sigmas, 
    const scalar_t* __restrict__ rgbs, 
    const scalar_t* __restrict__ ts, 
    scalar_t* weights_sum, scalar_t* depth, scalar_t* image
) {
    const uint32_t n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n >= n_alive) return;

    const int index = rays_alive[n]; // ray id
    
    // locate 
    sigmas += n * n_step;
    rgbs += n * n_step * 3;
    ts += n * n_step * 2;
    
    rays_t += index;
    weights_sum += index;
    depth += index;
    image += index * 3;

    float t;
    float d = depth[0], r = image[0], g = image[1], b = image[2], weight_sum = weights_sum[0];

    // accumulate 
    uint32_t step = 0;
    while (step < n_step) {
        
        // ray is terminated if t == 0
        if (ts[0] == 0) break;
        
        const float real_alpha = 1.0f - __expf(- sigmas[0] * ts[1]);
        const float alpha = binarize ? (real_alpha > 0.5 ? 1.0 : 0.0) : real_alpha;

        /* 
        T_0 = 1; T_i = \prod_{j=0}^{i-1} (1 - alpha_j)
        w_i = alpha_i * T_i
        --> 
        T_i = 1 - \sum_{j=0}^{i-1} w_j
        */
        const float T = 1 - weight_sum;
        const float weight = alpha * T;
        weight_sum += weight;

        t = ts[0];
        d += weight * t; // real depth
        r += weight * rgbs[0];
        g += weight * rgbs[1];
        b += weight * rgbs[2];

        //printf("[n=%d] num_steps=%d, alpha=%f, w=%f, T=%f, sum_dt=%f, d=%f\n", n, step, alpha, weight, T, sum_delta, d);

        // ray is terminated if T is too small
        // use a larger bound to further accelerate inference
        if (T < T_thresh) break;

        // locate
        sigmas++;
        rgbs += 3;
        ts += 2;
        step++;
    }

    //printf("[n=%d] rgb=(%f, %f, %f), d=%f\n", n, r, g, b, d);

    // rays_alive = -1 means ray is terminated early.
    if (step < n_step) {
        rays_alive[n] = -1;
    } else {
        rays_t[0] = t;
    }

    weights_sum[0] = weight_sum; // this is the thing I needed!
    depth[0] = d;
    image[0] = r;
    image[1] = g;
    image[2] = b;
}


void composite_rays(const uint32_t n_alive, const uint32_t n_step, const float T_thresh, const bool binarize, at::Tensor rays_alive, at::Tensor rays_t, at::Tensor sigmas, at::Tensor rgbs, at::Tensor ts, at::Tensor weights, at::Tensor depth, at::Tensor image) {
    static constexpr uint32_t N_THREAD = 128;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    image.scalar_type(), "composite_rays", ([&] {
        kernel_composite_rays<<<div_round_up(n_alive, N_THREAD), N_THREAD>>>(n_alive, n_step, T_thresh, binarize, rays_alive.data_ptr<int>(), rays_t.data_ptr<scalar_t>(), sigmas.data_ptr<scalar_t>(), rgbs.data_ptr<scalar_t>(), ts.data_ptr<scalar_t>(), weights.data_ptr<scalar_t>(), depth.data_ptr<scalar_t>(), image.data_ptr<scalar_t>());
    }));
}