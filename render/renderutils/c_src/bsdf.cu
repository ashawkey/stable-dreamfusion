#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related 
 * documentation and any modifications thereto. Any use, reproduction, 
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or 
 * its affiliates is strictly prohibited.
 */

#include "common.h"
#include "bsdf.h"

#define SPECULAR_EPSILON 1e-4f

//------------------------------------------------------------------------
// Lambert functions

__device__ inline float fwdLambert(const vec3f nrm, const vec3f wi)
{
    return max(dot(nrm, wi) / M_PI, 0.0f);
}

__device__ inline void bwdLambert(const vec3f nrm, const vec3f wi, vec3f& d_nrm, vec3f& d_wi, const float d_out)
{
    if (dot(nrm, wi) > 0.0f)
        bwdDot(nrm, wi, d_nrm, d_wi, d_out / M_PI);
}

//------------------------------------------------------------------------
// Fresnel Schlick 

__device__ inline float fwdFresnelSchlick(const float f0, const float f90, const float cosTheta)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float scale = powf(1.0f - _cosTheta, 5.0f);
    return f0 * (1.0f - scale) + f90 * scale;
}

__device__ inline void bwdFresnelSchlick(const float f0, const float f90, const float cosTheta, float& d_f0, float& d_f90, float& d_cosTheta, const float d_out)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float scale = pow(max(1.0f - _cosTheta, 0.0f), 5.0f);
    d_f0 += d_out * (1.0 - scale);
    d_f90 += d_out * scale;
    if (cosTheta >= SPECULAR_EPSILON && cosTheta < 1.0f - SPECULAR_EPSILON)
    {
        d_cosTheta += d_out * (f90 - f0) * -5.0f * powf(1.0f - cosTheta, 4.0f);
    }
}

__device__ inline vec3f fwdFresnelSchlick(const vec3f f0, const vec3f f90, const float cosTheta)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float scale = powf(1.0f - _cosTheta, 5.0f);
    return f0 * (1.0f - scale) + f90 * scale;
}

__device__ inline void bwdFresnelSchlick(const vec3f f0, const vec3f f90, const float cosTheta, vec3f& d_f0, vec3f& d_f90, float& d_cosTheta, const vec3f d_out)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float scale = pow(max(1.0f - _cosTheta, 0.0f), 5.0f);
    d_f0 += d_out * (1.0 - scale);
    d_f90 += d_out * scale;
    if (cosTheta >= SPECULAR_EPSILON && cosTheta < 1.0f - SPECULAR_EPSILON)
    {
        d_cosTheta += sum(d_out * (f90 - f0) * -5.0f * powf(1.0f - cosTheta, 4.0f));
    }
}

//------------------------------------------------------------------------
// Frostbite diffuse

__device__ inline float fwdFrostbiteDiffuse(const vec3f nrm, const vec3f wi, const vec3f wo, float linearRoughness)
{
    float wiDotN = dot(wi, nrm);
    float woDotN = dot(wo, nrm);
    if (wiDotN > 0.0f && woDotN > 0.0f)
    {
        vec3f h = safeNormalize(wo + wi);
        float wiDotH = dot(wi, h);

        float energyBias = 0.5f * linearRoughness;
        float energyFactor = 1.0f - (0.51f / 1.51f) * linearRoughness;
        float f90 = energyBias + 2.f * wiDotH * wiDotH * linearRoughness;
        float f0 = 1.f;
        
        float wiScatter = fwdFresnelSchlick(f0, f90, wiDotN);
        float woScatter = fwdFresnelSchlick(f0, f90, woDotN);
        
        return wiScatter * woScatter * energyFactor;
    }
    else return 0.0f;
}

__device__ inline void bwdFrostbiteDiffuse(const vec3f nrm, const vec3f wi, const vec3f wo, float linearRoughness, vec3f& d_nrm, vec3f& d_wi, vec3f& d_wo, float &d_linearRoughness, const float d_out)
{
    float wiDotN = dot(wi, nrm);
    float woDotN = dot(wo, nrm);

    if (wiDotN > 0.0f && woDotN > 0.0f)
    {
        vec3f h = safeNormalize(wo + wi);
        float wiDotH = dot(wi, h);

        float energyBias = 0.5f * linearRoughness;
        float energyFactor = 1.0f - (0.51f / 1.51f) * linearRoughness;
        float f90 = energyBias + 2.f * wiDotH * wiDotH * linearRoughness;
        float f0 = 1.f;
        
        float wiScatter = fwdFresnelSchlick(f0, f90, wiDotN);
        float woScatter = fwdFresnelSchlick(f0, f90, woDotN);

        // -------------- BWD --------------
        // Backprop: return wiScatter * woScatter * energyFactor;
        float d_wiScatter = d_out * woScatter * energyFactor;
        float d_woScatter = d_out * wiScatter * energyFactor;
        float d_energyFactor = d_out * wiScatter * woScatter; 

        // Backprop: float woScatter = fwdFresnelSchlick(f0, f90, woDotN);
        float d_woDotN = 0.0f, d_f0 = 0.0, d_f90 = 0.0f;
        bwdFresnelSchlick(f0, f90, woDotN, d_f0, d_f90, d_woDotN, d_woScatter);

        // Backprop: float wiScatter = fwdFresnelSchlick(fd0, fd90, wiDotN);
        float d_wiDotN = 0.0f;
        bwdFresnelSchlick(f0, f90, wiDotN, d_f0, d_f90, d_wiDotN, d_wiScatter);

        // Backprop: float f90 = energyBias + 2.f * wiDotH * wiDotH * linearRoughness;
        float d_energyBias = d_f90;
        float d_wiDotH = d_f90 * 4 * wiDotH * linearRoughness;
        d_linearRoughness += d_f90 * 2 * wiDotH * wiDotH;

        // Backprop: float energyFactor = 1.0f - (0.51f / 1.51f) * linearRoughness;
        d_linearRoughness -= (0.51f / 1.51f) * d_energyFactor;

        // Backprop: float energyBias = 0.5f * linearRoughness;
        d_linearRoughness += 0.5 * d_energyBias;

        // Backprop: float wiDotH = dot(wi, h);
        vec3f d_h(0);
        bwdDot(wi, h, d_wi, d_h, d_wiDotH);

        // Backprop: vec3f h = safeNormalize(wo + wi);     
        vec3f d_wo_wi(0);
        bwdSafeNormalize(wo + wi, d_wo_wi, d_h);
        d_wi += d_wo_wi; d_wo += d_wo_wi;

        bwdDot(wo, nrm, d_wo, d_nrm, d_woDotN);
        bwdDot(wi, nrm, d_wi, d_nrm, d_wiDotN);
    }
}

//------------------------------------------------------------------------
// Ndf GGX

__device__ inline float fwdNdfGGX(const float alphaSqr, const float cosTheta)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float d = (_cosTheta * alphaSqr - _cosTheta) * _cosTheta + 1.0f;
    return alphaSqr / (d * d * M_PI);
}

__device__ inline void bwdNdfGGX(const float alphaSqr, const float cosTheta, float& d_alphaSqr, float& d_cosTheta, const float d_out)
{
    // Torch only back propagates if clamp doesn't trigger
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float cosThetaSqr = _cosTheta * _cosTheta;
    d_alphaSqr += d_out * (1.0f - (alphaSqr + 1.0f) * cosThetaSqr) / (M_PI * powf((alphaSqr - 1.0) * cosThetaSqr + 1.0f, 3.0f));
    if (cosTheta > SPECULAR_EPSILON && cosTheta < 1.0f - SPECULAR_EPSILON)
    {
        d_cosTheta += d_out * -(4.0f * (alphaSqr - 1.0f) * alphaSqr * cosTheta) / (M_PI * powf((alphaSqr - 1.0) * cosThetaSqr + 1.0f, 3.0f));
    }
}

//------------------------------------------------------------------------
// Lambda GGX

__device__ inline float fwdLambdaGGX(const float alphaSqr, const float cosTheta)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float cosThetaSqr = _cosTheta * _cosTheta;
    float tanThetaSqr = (1.0 - cosThetaSqr) / cosThetaSqr;
    float res = 0.5f * (sqrtf(1.0f + alphaSqr * tanThetaSqr) - 1.0f);
    return res;
}

__device__ inline void bwdLambdaGGX(const float alphaSqr, const float cosTheta, float& d_alphaSqr, float& d_cosTheta, const float d_out)
{
    float _cosTheta = clamp(cosTheta, SPECULAR_EPSILON, 1.0f - SPECULAR_EPSILON);
    float cosThetaSqr = _cosTheta * _cosTheta;
    float tanThetaSqr = (1.0 - cosThetaSqr) / cosThetaSqr;
    float res = 0.5f * (sqrtf(1.0f + alphaSqr * tanThetaSqr) - 1.0f);

    d_alphaSqr += d_out * (0.25 * tanThetaSqr) / sqrtf(alphaSqr * tanThetaSqr + 1.0f);
    if (cosTheta > SPECULAR_EPSILON && cosTheta < 1.0f - SPECULAR_EPSILON)
        d_cosTheta += d_out * -(0.5 * alphaSqr) / (powf(_cosTheta, 3.0f) * sqrtf(alphaSqr / cosThetaSqr - alphaSqr + 1.0f));
}

//------------------------------------------------------------------------
// Masking GGX

__device__ inline float fwdMaskingSmithGGXCorrelated(const float alphaSqr, const float cosThetaI, const float cosThetaO)
{
    float lambdaI = fwdLambdaGGX(alphaSqr, cosThetaI);
    float lambdaO = fwdLambdaGGX(alphaSqr, cosThetaO);
    return 1.0f / (1.0f + lambdaI + lambdaO);
}

__device__ inline void bwdMaskingSmithGGXCorrelated(const float alphaSqr, const float cosThetaI, const float cosThetaO, float& d_alphaSqr, float& d_cosThetaI, float& d_cosThetaO, const float d_out)
{
    // FWD eval
    float lambdaI = fwdLambdaGGX(alphaSqr, cosThetaI);
    float lambdaO = fwdLambdaGGX(alphaSqr, cosThetaO);

    // BWD eval
    float d_lambdaIO = -d_out / powf(1.0f + lambdaI + lambdaO, 2.0f);
    bwdLambdaGGX(alphaSqr, cosThetaI, d_alphaSqr, d_cosThetaI, d_lambdaIO);
    bwdLambdaGGX(alphaSqr, cosThetaO, d_alphaSqr, d_cosThetaO, d_lambdaIO);
}

//------------------------------------------------------------------------
// GGX specular

__device__ vec3f fwdPbrSpecular(const vec3f col, const vec3f nrm, const vec3f wo, const vec3f wi, const float alpha, const float min_roughness)
{
    float _alpha = clamp(alpha, min_roughness * min_roughness, 1.0f);
    float alphaSqr = _alpha * _alpha;

    vec3f h = safeNormalize(wo + wi);
    float woDotN = dot(wo, nrm);
    float wiDotN = dot(wi, nrm);
    float woDotH = dot(wo, h);
    float nDotH = dot(nrm, h);

    float D = fwdNdfGGX(alphaSqr, nDotH);
    float G = fwdMaskingSmithGGXCorrelated(alphaSqr, woDotN, wiDotN);
    vec3f F = fwdFresnelSchlick(col, 1.0f, woDotH);
    vec3f w = F * D * G * 0.25 / woDotN;

    bool frontfacing = (woDotN > SPECULAR_EPSILON) & (wiDotN > SPECULAR_EPSILON);
    return frontfacing ? w : 0.0f;
}

__device__ void bwdPbrSpecular(
    const vec3f col, const vec3f nrm, const vec3f wo, const vec3f wi, const float alpha, const float min_roughness,
    vec3f& d_col, vec3f& d_nrm, vec3f& d_wo, vec3f& d_wi, float& d_alpha, const vec3f d_out)
{
    ///////////////////////////////////////////////////////////////////////
    // FWD eval

    float _alpha = clamp(alpha, min_roughness * min_roughness, 1.0f);
    float alphaSqr = _alpha * _alpha;

    vec3f h = safeNormalize(wo + wi);
    float woDotN = dot(wo, nrm);
    float wiDotN = dot(wi, nrm);
    float woDotH = dot(wo, h);
    float nDotH = dot(nrm, h);

    float D = fwdNdfGGX(alphaSqr, nDotH);
    float G = fwdMaskingSmithGGXCorrelated(alphaSqr, woDotN, wiDotN);
    vec3f F = fwdFresnelSchlick(col, 1.0f, woDotH);
    vec3f w = F * D * G * 0.25 / woDotN;
    bool frontfacing = (woDotN > SPECULAR_EPSILON) & (wiDotN > SPECULAR_EPSILON);

    if (frontfacing)
    {
        ///////////////////////////////////////////////////////////////////////
        // BWD eval

        vec3f d_F = d_out * D * G * 0.25f / woDotN;
        float d_D = sum(d_out * F * G * 0.25f / woDotN);
        float d_G = sum(d_out * F * D * 0.25f / woDotN);

        float d_woDotN = -sum(d_out * F * D * G * 0.25f / (woDotN * woDotN));

        vec3f d_f90(0);
        float d_woDotH(0), d_wiDotN(0), d_nDotH(0), d_alphaSqr(0);
        bwdFresnelSchlick(col, 1.0f, woDotH, d_col, d_f90, d_woDotH, d_F);
        bwdMaskingSmithGGXCorrelated(alphaSqr, woDotN, wiDotN, d_alphaSqr, d_woDotN, d_wiDotN, d_G);
        bwdNdfGGX(alphaSqr, nDotH, d_alphaSqr, d_nDotH, d_D);

        vec3f d_h(0);
        bwdDot(nrm, h, d_nrm, d_h, d_nDotH);
        bwdDot(wo, h, d_wo, d_h, d_woDotH);
        bwdDot(wi, nrm, d_wi, d_nrm, d_wiDotN);
        bwdDot(wo, nrm, d_wo, d_nrm, d_woDotN);

        vec3f d_h_unnorm(0);
        bwdSafeNormalize(wo + wi, d_h_unnorm, d_h);
        d_wo += d_h_unnorm;
        d_wi += d_h_unnorm;

        if (alpha > min_roughness * min_roughness)
            d_alpha += d_alphaSqr * 2 * alpha;
    }
}

//------------------------------------------------------------------------
// Full PBR BSDF

__device__ vec3f fwdPbrBSDF(const vec3f kd, const vec3f arm, const vec3f pos, const vec3f nrm, const vec3f view_pos, const vec3f light_pos, const float min_roughness, int BSDF)
{
    vec3f wo = safeNormalize(view_pos - pos);
    vec3f wi = safeNormalize(light_pos - pos);

    float alpha = arm.y * arm.y;
    vec3f spec_col = (0.04f * (1.0f - arm.z) + kd * arm.z) * (1.0 - arm.x);
    vec3f diff_col = kd * (1.0f - arm.z);

    float diff = 0.0f;
    if (BSDF == 0)
        diff = fwdLambert(nrm, wi);
    else
        diff = fwdFrostbiteDiffuse(nrm, wi, wo, arm.y);    
    vec3f diffuse = diff_col * diff;
    vec3f specular = fwdPbrSpecular(spec_col, nrm, wo, wi, alpha, min_roughness);

    return diffuse + specular;
}

__device__ void bwdPbrBSDF(
    const vec3f kd, const vec3f arm, const vec3f pos, const vec3f nrm, const vec3f view_pos, const vec3f light_pos, const float min_roughness, int BSDF,
    vec3f& d_kd, vec3f& d_arm, vec3f& d_pos, vec3f& d_nrm, vec3f& d_view_pos, vec3f& d_light_pos, const vec3f d_out)
{
    ////////////////////////////////////////////////////////////////////////
    // FWD
    vec3f _wi = light_pos - pos;
    vec3f _wo = view_pos - pos;
    vec3f wi = safeNormalize(_wi);
    vec3f wo = safeNormalize(_wo);

    float alpha = arm.y * arm.y;
    vec3f spec_col = (0.04f * (1.0f - arm.z) + kd * arm.z) * (1.0 - arm.x);
    vec3f diff_col = kd * (1.0f - arm.z);
    float diff = 0.0f;
    if (BSDF == 0)
        diff = fwdLambert(nrm, wi);
    else
        diff = fwdFrostbiteDiffuse(nrm, wi, wo, arm.y);    

    ////////////////////////////////////////////////////////////////////////
    // BWD

    float d_alpha(0);
    vec3f d_spec_col(0), d_wi(0), d_wo(0);
    bwdPbrSpecular(spec_col, nrm, wo, wi, alpha, min_roughness, d_spec_col, d_nrm, d_wo, d_wi, d_alpha, d_out);

    float d_diff = sum(diff_col * d_out);
    if (BSDF == 0)
        bwdLambert(nrm, wi, d_nrm, d_wi, d_diff);
    else
        bwdFrostbiteDiffuse(nrm, wi, wo, arm.y, d_nrm, d_wi, d_wo, d_arm.y, d_diff);    

    // Backprop: diff_col = kd * (1.0f - arm.z)
    vec3f d_diff_col = d_out * diff;
    d_kd += d_diff_col * (1.0f - arm.z);
    d_arm.z -= sum(d_diff_col * kd);

    // Backprop: spec_col = (0.04f * (1.0f - arm.z) + kd * arm.z) * (1.0 - arm.x)
    d_kd -= d_spec_col * (arm.x - 1.0f) * arm.z;
    d_arm.x += sum(d_spec_col * (arm.z * (0.04f - kd) - 0.04f));
    d_arm.z -= sum(d_spec_col * (kd - 0.04f) * (arm.x - 1.0f));

    // Backprop: alpha = arm.y * arm.y
    d_arm.y += d_alpha * 2 * arm.y;

    // Backprop: vec3f wi = safeNormalize(light_pos - pos);
    vec3f d__wi(0);
    bwdSafeNormalize(_wi, d__wi, d_wi);
    d_light_pos += d__wi;
    d_pos -= d__wi;

    // Backprop: vec3f wo = safeNormalize(view_pos - pos);
    vec3f d__wo(0);
    bwdSafeNormalize(_wo, d__wo, d_wo);
    d_view_pos += d__wo;
    d_pos -= d__wo;
}

//------------------------------------------------------------------------
// Kernels

__global__ void LambertFwdKernel(LambertKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);

    float res = fwdLambert(nrm, wi);

    p.out.store(px, py, pz, res);
}

__global__ void LambertBwdKernel(LambertKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);
    float d_out = p.out.fetch1(px, py, pz);

    vec3f d_nrm(0), d_wi(0);
    bwdLambert(nrm, wi, d_nrm, d_wi, d_out);

    p.nrm.store_grad(px, py, pz, d_nrm);
    p.wi.store_grad(px, py, pz, d_wi);
}

__global__ void FrostbiteDiffuseFwdKernel(FrostbiteDiffuseKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);
    vec3f wo = p.wo.fetch3(px, py, pz);
    float linearRoughness = p.linearRoughness.fetch1(px, py, pz);

    float res = fwdFrostbiteDiffuse(nrm, wi, wo, linearRoughness);

    p.out.store(px, py, pz, res);
}

__global__ void FrostbiteDiffuseBwdKernel(FrostbiteDiffuseKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);
    vec3f wo = p.wo.fetch3(px, py, pz);
    float linearRoughness = p.linearRoughness.fetch1(px, py, pz);
    float d_out = p.out.fetch1(px, py, pz);

    float d_linearRoughness = 0.0f;
    vec3f d_nrm(0), d_wi(0), d_wo(0);
    bwdFrostbiteDiffuse(nrm, wi, wo, linearRoughness, d_nrm, d_wi, d_wo, d_linearRoughness, d_out);

    p.nrm.store_grad(px, py, pz, d_nrm);
    p.wi.store_grad(px, py, pz, d_wi);
    p.wo.store_grad(px, py, pz, d_wo);
    p.linearRoughness.store_grad(px, py, pz, d_linearRoughness);
}

__global__ void FresnelShlickFwdKernel(FresnelShlickKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f f0 = p.f0.fetch3(px, py, pz);
    vec3f f90 = p.f90.fetch3(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);

    vec3f res = fwdFresnelSchlick(f0, f90, cosTheta);
    p.out.store(px, py, pz, res);
}

__global__ void FresnelShlickBwdKernel(FresnelShlickKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f f0 = p.f0.fetch3(px, py, pz);
    vec3f f90 = p.f90.fetch3(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);
    vec3f d_out = p.out.fetch3(px, py, pz);

    vec3f d_f0(0), d_f90(0);
    float d_cosTheta(0);
    bwdFresnelSchlick(f0, f90, cosTheta, d_f0, d_f90, d_cosTheta, d_out);

    p.f0.store_grad(px, py, pz, d_f0);
    p.f90.store_grad(px, py, pz, d_f90);
    p.cosTheta.store_grad(px, py, pz, d_cosTheta);
}

__global__ void ndfGGXFwdKernel(NdfGGXParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);
    float res = fwdNdfGGX(alphaSqr, cosTheta);
    
    p.out.store(px, py, pz, res);
}

__global__ void ndfGGXBwdKernel(NdfGGXParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);
    float d_out = p.out.fetch1(px, py, pz);

    float d_alphaSqr(0), d_cosTheta(0);
    bwdNdfGGX(alphaSqr, cosTheta, d_alphaSqr, d_cosTheta, d_out);

    p.alphaSqr.store_grad(px, py, pz, d_alphaSqr);
    p.cosTheta.store_grad(px, py, pz, d_cosTheta);
}

__global__ void lambdaGGXFwdKernel(NdfGGXParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);
    float res = fwdLambdaGGX(alphaSqr, cosTheta);

    p.out.store(px, py, pz, res);
}

__global__ void lambdaGGXBwdKernel(NdfGGXParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosTheta = p.cosTheta.fetch1(px, py, pz);
    float d_out = p.out.fetch1(px, py, pz);

    float d_alphaSqr(0), d_cosTheta(0);
    bwdLambdaGGX(alphaSqr, cosTheta, d_alphaSqr, d_cosTheta, d_out);

    p.alphaSqr.store_grad(px, py, pz, d_alphaSqr);
    p.cosTheta.store_grad(px, py, pz, d_cosTheta);
}

__global__ void maskingSmithFwdKernel(MaskingSmithParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosThetaI = p.cosThetaI.fetch1(px, py, pz);
    float cosThetaO = p.cosThetaO.fetch1(px, py, pz);
    float res = fwdMaskingSmithGGXCorrelated(alphaSqr, cosThetaI, cosThetaO);
    
    p.out.store(px, py, pz, res);
}

__global__ void maskingSmithBwdKernel(MaskingSmithParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    float alphaSqr = p.alphaSqr.fetch1(px, py, pz);
    float cosThetaI = p.cosThetaI.fetch1(px, py, pz);
    float cosThetaO = p.cosThetaO.fetch1(px, py, pz);
    float d_out = p.out.fetch1(px, py, pz);

    float d_alphaSqr(0), d_cosThetaI(0), d_cosThetaO(0);
    bwdMaskingSmithGGXCorrelated(alphaSqr, cosThetaI, cosThetaO, d_alphaSqr, d_cosThetaI, d_cosThetaO, d_out);

    p.alphaSqr.store_grad(px, py, pz, d_alphaSqr);
    p.cosThetaI.store_grad(px, py, pz, d_cosThetaI);
    p.cosThetaO.store_grad(px, py, pz, d_cosThetaO);
}

__global__ void pbrSpecularFwdKernel(PbrSpecular p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f col = p.col.fetch3(px, py, pz);
    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wo = p.wo.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);
    float alpha = p.alpha.fetch1(px, py, pz);

    vec3f res = fwdPbrSpecular(col, nrm, wo, wi, alpha, p.min_roughness);

    p.out.store(px, py, pz, res);
}

__global__ void pbrSpecularBwdKernel(PbrSpecular p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f col = p.col.fetch3(px, py, pz);
    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f wo = p.wo.fetch3(px, py, pz);
    vec3f wi = p.wi.fetch3(px, py, pz);
    float alpha = p.alpha.fetch1(px, py, pz);
    vec3f d_out = p.out.fetch3(px, py, pz);

    float d_alpha(0);
    vec3f d_col(0), d_nrm(0), d_wo(0), d_wi(0);
    bwdPbrSpecular(col, nrm, wo, wi, alpha, p.min_roughness, d_col, d_nrm, d_wo, d_wi, d_alpha, d_out);

    p.col.store_grad(px, py, pz, d_col);
    p.nrm.store_grad(px, py, pz, d_nrm);
    p.wo.store_grad(px, py, pz, d_wo);
    p.wi.store_grad(px, py, pz, d_wi);
    p.alpha.store_grad(px, py, pz, d_alpha);
}

__global__ void pbrBSDFFwdKernel(PbrBSDF p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f kd = p.kd.fetch3(px, py, pz);
    vec3f arm = p.arm.fetch3(px, py, pz);
    vec3f pos = p.pos.fetch3(px, py, pz);
    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f view_pos = p.view_pos.fetch3(px, py, pz);
    vec3f light_pos = p.light_pos.fetch3(px, py, pz);

    vec3f res = fwdPbrBSDF(kd, arm, pos, nrm, view_pos, light_pos, p.min_roughness, p.BSDF);

    p.out.store(px, py, pz, res);
}
__global__ void pbrBSDFBwdKernel(PbrBSDF p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f kd = p.kd.fetch3(px, py, pz);
    vec3f arm = p.arm.fetch3(px, py, pz);
    vec3f pos = p.pos.fetch3(px, py, pz);
    vec3f nrm = p.nrm.fetch3(px, py, pz);
    vec3f view_pos = p.view_pos.fetch3(px, py, pz);
    vec3f light_pos = p.light_pos.fetch3(px, py, pz);
    vec3f d_out = p.out.fetch3(px, py, pz);

    vec3f d_kd(0), d_arm(0), d_pos(0), d_nrm(0), d_view_pos(0), d_light_pos(0);
    bwdPbrBSDF(kd, arm, pos, nrm, view_pos, light_pos, p.min_roughness, p.BSDF, d_kd, d_arm, d_pos, d_nrm, d_view_pos, d_light_pos, d_out);

    p.kd.store_grad(px, py, pz, d_kd);
    p.arm.store_grad(px, py, pz, d_arm);
    p.pos.store_grad(px, py, pz, d_pos);
    p.nrm.store_grad(px, py, pz, d_nrm);
    p.view_pos.store_grad(px, py, pz, d_view_pos);
    p.light_pos.store_grad(px, py, pz, d_light_pos);
}
