#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related 
 * documentation and any modifications thereto. Any use, reproduction, 
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or 
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "common.h"
#include "mesh.h"


//------------------------------------------------------------------------
// Kernels

__global__ void xfmPointsFwdKernel(XfmKernelParams p)
{
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    __shared__ float mtx[4][4];
    if (threadIdx.x < 16)
        mtx[threadIdx.x % 4][threadIdx.x / 4] = p.matrix.fetch(p.matrix.nhwcIndex(pz, threadIdx.x / 4, threadIdx.x % 4, 0));
    __syncthreads();
    
    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    if (p.isPoints)
    {
        p.out.store(p.out.nhwcIndex(pz, px, 0, 0), pos.x * mtx[0][0] + pos.y * mtx[1][0] + pos.z * mtx[2][0] + mtx[3][0]);
        p.out.store(p.out.nhwcIndex(pz, px, 1, 0), pos.x * mtx[0][1] + pos.y * mtx[1][1] + pos.z * mtx[2][1] + mtx[3][1]);
        p.out.store(p.out.nhwcIndex(pz, px, 2, 0), pos.x * mtx[0][2] + pos.y * mtx[1][2] + pos.z * mtx[2][2] + mtx[3][2]);
        p.out.store(p.out.nhwcIndex(pz, px, 3, 0), pos.x * mtx[0][3] + pos.y * mtx[1][3] + pos.z * mtx[2][3] + mtx[3][3]);
    }
    else
    {
        p.out.store(p.out.nhwcIndex(pz, px, 0, 0), pos.x * mtx[0][0] + pos.y * mtx[1][0] + pos.z * mtx[2][0]);
        p.out.store(p.out.nhwcIndex(pz, px, 1, 0), pos.x * mtx[0][1] + pos.y * mtx[1][1] + pos.z * mtx[2][1]);
        p.out.store(p.out.nhwcIndex(pz, px, 2, 0), pos.x * mtx[0][2] + pos.y * mtx[1][2] + pos.z * mtx[2][2]);
    }
}

__global__ void xfmPointsBwdKernel(XfmKernelParams p)
{ 
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int pz = blockIdx.z * blockDim.z + threadIdx.z;

    __shared__ float mtx[4][4];
    if (threadIdx.x < 16)
        mtx[threadIdx.x % 4][threadIdx.x / 4] = p.matrix.fetch(p.matrix.nhwcIndex(pz, threadIdx.x / 4, threadIdx.x % 4, 0));
    __syncthreads();

    if (px >= p.gridSize.x)
        return;

    vec3f pos(
        p.points.fetch(p.points.nhwcIndex(pz, px, 0, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 1, 0)),
        p.points.fetch(p.points.nhwcIndex(pz, px, 2, 0))
    );

    vec4f d_out(
        p.out.fetch(p.out.nhwcIndex(pz, px, 0, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 1, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 2, 0)),
        p.out.fetch(p.out.nhwcIndex(pz, px, 3, 0))
    );

    if (p.isPoints)
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2] + d_out.w * mtx[0][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2] + d_out.w * mtx[1][3]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2] + d_out.w * mtx[2][3]);
    }
    else
    {
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 0, 0), d_out.x * mtx[0][0] + d_out.y * mtx[0][1] + d_out.z * mtx[0][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 1, 0), d_out.x * mtx[1][0] + d_out.y * mtx[1][1] + d_out.z * mtx[1][2]);
        p.points.store_grad(p.points.nhwcIndexContinuous(pz, px, 2, 0), d_out.x * mtx[2][0] + d_out.y * mtx[2][1] + d_out.z * mtx[2][2]);
    }
}