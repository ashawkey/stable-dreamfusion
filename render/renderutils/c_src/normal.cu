#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related 
 * documentation and any modifications thereto. Any use, reproduction, 
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or 
 * its affiliates is strictly prohibited.
 */

#include "common.h"
#include "normal.h"

#define NORMAL_THRESHOLD 0.1f

//------------------------------------------------------------------------
// Perturb shading normal by tangent frame

__device__ vec3f fwdPerturbNormal(const vec3f perturbed_nrm, const vec3f smooth_nrm, const vec3f smooth_tng, bool opengl)
{
    vec3f _smooth_bitng = cross(smooth_tng, smooth_nrm);
    vec3f smooth_bitng = safeNormalize(_smooth_bitng);
    vec3f _shading_nrm = smooth_tng * perturbed_nrm.x + (opengl ? -1 : 1) * smooth_bitng * perturbed_nrm.y + smooth_nrm * max(perturbed_nrm.z, 0.0f);
    return safeNormalize(_shading_nrm);
}

__device__ void bwdPerturbNormal(const vec3f perturbed_nrm, const vec3f smooth_nrm, const vec3f smooth_tng, vec3f &d_perturbed_nrm, vec3f &d_smooth_nrm, vec3f &d_smooth_tng, const vec3f d_out, bool opengl)
{
    ////////////////////////////////////////////////////////////////////////
    // FWD
    vec3f _smooth_bitng = cross(smooth_tng, smooth_nrm);
    vec3f smooth_bitng = safeNormalize(_smooth_bitng);
    vec3f _shading_nrm = smooth_tng * perturbed_nrm.x + (opengl ? -1 : 1) * smooth_bitng * perturbed_nrm.y + smooth_nrm * max(perturbed_nrm.z, 0.0f);
        
    ////////////////////////////////////////////////////////////////////////
    // BWD
    vec3f d_shading_nrm(0);
    bwdSafeNormalize(_shading_nrm, d_shading_nrm, d_out);

    vec3f d_smooth_bitng(0);
    
    if (perturbed_nrm.z > 0.0f)
    {
        d_smooth_nrm += d_shading_nrm * perturbed_nrm.z;
        d_perturbed_nrm.z += sum(d_shading_nrm * smooth_nrm);
    }

    d_smooth_bitng += (opengl ? -1 : 1) * d_shading_nrm * perturbed_nrm.y;
    d_perturbed_nrm.y += (opengl ? -1 : 1) * sum(d_shading_nrm * smooth_bitng);

    d_smooth_tng += d_shading_nrm * perturbed_nrm.x;
    d_perturbed_nrm.x += sum(d_shading_nrm * smooth_tng);

    vec3f d__smooth_bitng(0);
    bwdSafeNormalize(_smooth_bitng, d__smooth_bitng, d_smooth_bitng);

    bwdCross(smooth_tng, smooth_nrm, d_smooth_tng, d_smooth_nrm, d__smooth_bitng);
}

//------------------------------------------------------------------------
#define bent_nrm_eps 0.001f

__device__ vec3f fwdBendNormal(const vec3f view_vec, const vec3f smooth_nrm, const vec3f geom_nrm)
{
    float dp = dot(view_vec, smooth_nrm);
    float t = clamp(dp / NORMAL_THRESHOLD, 0.0f, 1.0f);
    return geom_nrm * (1.0f - t) + smooth_nrm * t;
}

__device__ void bwdBendNormal(const vec3f view_vec, const vec3f smooth_nrm, const vec3f geom_nrm, vec3f& d_view_vec, vec3f& d_smooth_nrm, vec3f& d_geom_nrm, const vec3f d_out)
{
    ////////////////////////////////////////////////////////////////////////
    // FWD
    float dp = dot(view_vec, smooth_nrm);
    float t = clamp(dp / NORMAL_THRESHOLD, 0.0f, 1.0f);

    ////////////////////////////////////////////////////////////////////////
    // BWD
    if (dp > NORMAL_THRESHOLD)
        d_smooth_nrm += d_out;
    else
    {
        // geom_nrm * (1.0f - t) + smooth_nrm * t;
        d_geom_nrm   += d_out * (1.0f - t);
        d_smooth_nrm += d_out * t;
        float d_t = sum(d_out * (smooth_nrm - geom_nrm));

        float d_dp = dp < 0.0f || dp > NORMAL_THRESHOLD ? 0.0f : d_t / NORMAL_THRESHOLD;

        bwdDot(view_vec, smooth_nrm, d_view_vec, d_smooth_nrm, d_dp);
    }
}

//------------------------------------------------------------------------
// Kernels

__global__ void PrepareShadingNormalFwdKernel(PrepareShadingNormalKernelParams p) 
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f pos = p.pos.fetch3(px, py, pz);
    vec3f view_pos = p.view_pos.fetch3(px, py, pz);
    vec3f perturbed_nrm = p.perturbed_nrm.fetch3(px, py, pz);
    vec3f _smooth_nrm = p.smooth_nrm.fetch3(px, py, pz);
    vec3f _smooth_tng = p.smooth_tng.fetch3(px, py, pz);
    vec3f geom_nrm = p.geom_nrm.fetch3(px, py, pz);

    vec3f smooth_nrm = safeNormalize(_smooth_nrm);
    vec3f smooth_tng = safeNormalize(_smooth_tng);
    vec3f view_vec = safeNormalize(view_pos - pos);
    vec3f shading_nrm = fwdPerturbNormal(perturbed_nrm, smooth_nrm, smooth_tng, p.opengl);

    vec3f res;
    if (p.two_sided_shading && dot(view_vec, geom_nrm) < 0.0f)
        res = fwdBendNormal(view_vec, -shading_nrm, -geom_nrm);
    else
        res = fwdBendNormal(view_vec, shading_nrm, geom_nrm);

    p.out.store(px, py, pz, res);
}

__global__ void PrepareShadingNormalBwdKernel(PrepareShadingNormalKernelParams p) 
{ 
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    vec3f pos = p.pos.fetch3(px, py, pz);
    vec3f view_pos = p.view_pos.fetch3(px, py, pz);
    vec3f perturbed_nrm = p.perturbed_nrm.fetch3(px, py, pz);
    vec3f _smooth_nrm = p.smooth_nrm.fetch3(px, py, pz);
    vec3f _smooth_tng = p.smooth_tng.fetch3(px, py, pz);
    vec3f geom_nrm = p.geom_nrm.fetch3(px, py, pz);
    vec3f d_out = p.out.fetch3(px, py, pz);

    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // FWD

    vec3f smooth_nrm = safeNormalize(_smooth_nrm);
    vec3f smooth_tng = safeNormalize(_smooth_tng);
    vec3f _view_vec = view_pos - pos;
    vec3f view_vec = safeNormalize(view_pos - pos);

    vec3f shading_nrm = fwdPerturbNormal(perturbed_nrm, smooth_nrm, smooth_tng, p.opengl);

    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // BWD

    vec3f d_view_vec(0), d_shading_nrm(0), d_geom_nrm(0);
    if (p.two_sided_shading && dot(view_vec, geom_nrm) < 0.0f)
    {
        bwdBendNormal(view_vec, -shading_nrm, -geom_nrm, d_view_vec, d_shading_nrm, d_geom_nrm, d_out);
        d_shading_nrm = -d_shading_nrm;
        d_geom_nrm = -d_geom_nrm;
    }
    else
        bwdBendNormal(view_vec, shading_nrm, geom_nrm, d_view_vec, d_shading_nrm, d_geom_nrm, d_out);

    vec3f d_perturbed_nrm(0), d_smooth_nrm(0), d_smooth_tng(0);
    bwdPerturbNormal(perturbed_nrm, smooth_nrm, smooth_tng, d_perturbed_nrm, d_smooth_nrm, d_smooth_tng, d_shading_nrm, p.opengl);

    vec3f d__view_vec(0), d__smooth_nrm(0), d__smooth_tng(0);
    bwdSafeNormalize(_view_vec, d__view_vec, d_view_vec);
    bwdSafeNormalize(_smooth_nrm, d__smooth_nrm, d_smooth_nrm);
    bwdSafeNormalize(_smooth_tng, d__smooth_tng, d_smooth_tng);

    p.pos.store_grad(px, py, pz, -d__view_vec);
    p.view_pos.store_grad(px, py, pz, d__view_vec);
    p.perturbed_nrm.store_grad(px, py, pz, d_perturbed_nrm);
    p.smooth_nrm.store_grad(px, py, pz, d__smooth_nrm);
    p.smooth_tng.store_grad(px, py, pz, d__smooth_tng);
    p.geom_nrm.store_grad(px, py, pz, d_geom_nrm);
}