#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related 
 * documentation and any modifications thereto. Any use, reproduction, 
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or 
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_runtime.h>

#include "common.h"
#include "loss.h"

//------------------------------------------------------------------------
// Utils

__device__ inline float bwdAbs(float x) { return x == 0.0f ? 0.0f : x < 0.0f ? -1.0f : 1.0f; }

__device__ float warpSum(float val) {
    for (int i = 1; i < 32; i *= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, i);
    return val;
}

//------------------------------------------------------------------------
// Tonemapping

__device__ inline float fwdSRGB(float x)
{
    return x > 0.0031308f ? powf(max(x, 0.0031308f), 1.0f / 2.4f) * 1.055f - 0.055f : 12.92f * max(x, 0.0f);
}

__device__ inline void bwdSRGB(float x, float &d_x, float d_out)
{
    if (x > 0.0031308f)
        d_x += d_out * 0.439583f / powf(x, 0.583333f);
    else if (x > 0.0f)
        d_x += d_out * 12.92f;
}

__device__ inline vec3f fwdTonemapLogSRGB(vec3f x)
{
    return vec3f(fwdSRGB(logf(x.x + 1.0f)), fwdSRGB(logf(x.y + 1.0f)), fwdSRGB(logf(x.z + 1.0f)));
}

__device__ inline void bwdTonemapLogSRGB(vec3f x, vec3f& d_x, vec3f d_out)
{
    if (x.x > 0.0f && x.x < 65535.0f)
    {
        bwdSRGB(logf(x.x + 1.0f), d_x.x, d_out.x);
        d_x.x *= 1 / (x.x + 1.0f);
    }
    if (x.y > 0.0f && x.y < 65535.0f)
    {
        bwdSRGB(logf(x.y + 1.0f), d_x.y, d_out.y);
        d_x.y *= 1 / (x.y + 1.0f);
    }
    if (x.z > 0.0f && x.z < 65535.0f)
    {
        bwdSRGB(logf(x.z + 1.0f), d_x.z, d_out.z);
        d_x.z *= 1 / (x.z + 1.0f);
    }
}

__device__ inline float fwdRELMSE(float img, float target, float eps = 0.1f)
{
    return (img - target) * (img - target) / (img * img + target * target + eps);
}

__device__ inline void bwdRELMSE(float img, float target, float &d_img, float &d_target, float d_out, float eps = 0.1f)
{
    float denom  = (target * target + img * img + eps);
    d_img    += d_out * 2 * (img - target) * (target * (target + img) + eps) / (denom * denom);
    d_target -= d_out * 2 * (img - target) * (img * (target + img) + eps) / (denom * denom);
}

__device__ inline float fwdSMAPE(float img, float target, float eps=0.01f)
{
    return abs(img - target) / (img + target + eps);
}

__device__ inline void bwdSMAPE(float img, float target, float& d_img, float& d_target, float d_out, float eps = 0.01f)
{
    float denom = (target + img + eps);
    d_img    += d_out * bwdAbs(img - target) * (2 * target + eps) / (denom * denom);
    d_target -= d_out * bwdAbs(img - target) * (2 * img + eps) / (denom * denom);
}

//------------------------------------------------------------------------
// Kernels

__global__ void imgLossFwdKernel(LossKernelParams p)
{
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;

    float floss = 0.0f;
    if (px < p.gridSize.x && py < p.gridSize.y && pz < p.gridSize.z)
    {
        vec3f img = p.img.fetch3(px, py, pz);
        vec3f target = p.target.fetch3(px, py, pz);

        img = vec3f(clamp(img.x, 0.0f, 65535.0f), clamp(img.y, 0.0f, 65535.0f), clamp(img.z, 0.0f, 65535.0f));
        target = vec3f(clamp(target.x, 0.0f, 65535.0f), clamp(target.y, 0.0f, 65535.0f), clamp(target.z, 0.0f, 65535.0f));

        if (p.tonemapper == TONEMAPPER_LOG_SRGB)
        {
            img = fwdTonemapLogSRGB(img);
            target = fwdTonemapLogSRGB(target);
        }

        vec3f vloss(0);
        if (p.loss == LOSS_MSE)
            vloss = (img - target) * (img - target);
        else if (p.loss == LOSS_RELMSE)
            vloss = vec3f(fwdRELMSE(img.x, target.x), fwdRELMSE(img.y, target.y), fwdRELMSE(img.z, target.z));
        else if (p.loss == LOSS_SMAPE)
            vloss = vec3f(fwdSMAPE(img.x, target.x), fwdSMAPE(img.y, target.y), fwdSMAPE(img.z, target.z));
        else
            vloss = vec3f(abs(img.x - target.x), abs(img.y - target.y), abs(img.z - target.z));
        
        floss = sum(vloss) / 3.0f;
    }

    floss = warpSum(floss);

    dim3 warpSize = getWarpSize(blockDim);
    if (px < p.gridSize.x && py < p.gridSize.y && pz < p.gridSize.z && threadIdx.x % warpSize.x == 0 && threadIdx.y % warpSize.y == 0 && threadIdx.z % warpSize.z == 0)
        p.out.store(px / warpSize.x, py / warpSize.y, pz / warpSize.z, floss);
}

__global__ void imgLossBwdKernel(LossKernelParams p)
{ 
    // Calculate pixel position.
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int pz = blockIdx.z;

    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    dim3 warpSize = getWarpSize(blockDim);

    vec3f _img = p.img.fetch3(px, py, pz);
    vec3f _target = p.target.fetch3(px, py, pz);
    float d_out = p.out.fetch1(px / warpSize.x, py / warpSize.y, pz / warpSize.z);

    /////////////////////////////////////////////////////////////////////
    // FWD

    vec3f img = _img, target = _target;
    if (p.tonemapper == TONEMAPPER_LOG_SRGB)
    {
        img = fwdTonemapLogSRGB(img);
        target = fwdTonemapLogSRGB(target);
    }

    /////////////////////////////////////////////////////////////////////
    // BWD

    vec3f d_vloss = vec3f(d_out, d_out, d_out) / 3.0f;

    vec3f d_img(0), d_target(0);
    if (p.loss == LOSS_MSE)
    {
        d_img = vec3f(d_vloss.x * 2 * (img.x - target.x), d_vloss.y * 2 * (img.y - target.y), d_vloss.x * 2 * (img.z - target.z));
        d_target = -d_img;
    }
    else if (p.loss == LOSS_RELMSE)
    {
        bwdRELMSE(img.x, target.x, d_img.x, d_target.x, d_vloss.x);
        bwdRELMSE(img.y, target.y, d_img.y, d_target.y, d_vloss.y);
        bwdRELMSE(img.z, target.z, d_img.z, d_target.z, d_vloss.z);
    }
    else if (p.loss == LOSS_SMAPE)
    {
        bwdSMAPE(img.x, target.x, d_img.x, d_target.x, d_vloss.x);
        bwdSMAPE(img.y, target.y, d_img.y, d_target.y, d_vloss.y);
        bwdSMAPE(img.z, target.z, d_img.z, d_target.z, d_vloss.z);
    }
    else
    {
        d_img = d_vloss * vec3f(bwdAbs(img.x - target.x), bwdAbs(img.y - target.y), bwdAbs(img.z - target.z));
        d_target = -d_img;
    }


    if (p.tonemapper == TONEMAPPER_LOG_SRGB)
    {
        vec3f d__img(0), d__target(0);
        bwdTonemapLogSRGB(_img, d__img, d_img);
        bwdTonemapLogSRGB(_target, d__target, d_target);
        d_img = d__img; d_target = d__target;
    }

    if (_img.x <= 0.0f || _img.x >= 65535.0f) d_img.x = 0;
    if (_img.y <= 0.0f || _img.y >= 65535.0f) d_img.y = 0;
    if (_img.z <= 0.0f || _img.z >= 65535.0f) d_img.z = 0;
    if (_target.x <= 0.0f || _target.x >= 65535.0f) d_target.x = 0;
    if (_target.y <= 0.0f || _target.y >= 65535.0f) d_target.y = 0;
    if (_target.z <= 0.0f || _target.z >= 65535.0f) d_target.z = 0;

    p.img.store_grad(px, py, pz, d_img);
    p.target.store_grad(px, py, pz, d_target);
}