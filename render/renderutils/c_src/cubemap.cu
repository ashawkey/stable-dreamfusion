#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related 
 * documentation and any modifications thereto. Any use, reproduction, 
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or 
 * its affiliates is strictly prohibited.
 */

#include "common.h"
#include "cubemap.h"
#include <float.h>

// https://cgvr.cs.uni-bremen.de/teaching/cg_literatur/Spherical,%20Cubic,%20and%20Parabolic%20Environment%20Mappings.pdf
__device__ float pixel_area(int x, int y, int N)
{
    if (N > 1)
    {
        int H = N / 2;
        x = abs(x - H);
        y = abs(y - H);
        float dx = atan((float)(x + 1) / (float)H) - atan((float)x / (float)H);
        float dy = atan((float)(y + 1) / (float)H) - atan((float)y / (float)H);
        return dx * dy;
    }
    else
        return 1;
}

__device__ vec3f cube_to_dir(int x, int y, int side, int N)
{
    float fx = 2.0f * (((float)x + 0.5f) / (float)N) - 1.0f;
    float fy = 2.0f * (((float)y + 0.5f) / (float)N) - 1.0f;
    switch (side)
    {
        case 0: return safeNormalize(vec3f(1, -fy, -fx));
        case 1: return safeNormalize(vec3f(-1, -fy, fx));
        case 2: return safeNormalize(vec3f(fx, 1, fy));
        case 3: return safeNormalize(vec3f(fx, -1, -fy));
        case 4: return safeNormalize(vec3f(fx, -fy, 1));
        case 5: return safeNormalize(vec3f(-fx, -fy, -1));
    }
    return vec3f(0,0,0); // Unreachable
}

__device__ vec3f dir_to_side(int side, vec3f v)
{
    switch (side)
    {
    case 0: return vec3f(-v.z, -v.y,  v.x);
    case 1: return vec3f( v.z, -v.y, -v.x);
    case 2: return vec3f( v.x,  v.z,  v.y);
    case 3: return vec3f( v.x, -v.z, -v.y);
    case 4: return vec3f( v.x, -v.y,  v.z);
    case 5: return vec3f(-v.x, -v.y, -v.z);
    }
    return vec3f(0,0,0); // Unreachable
}

__device__ void extents_1d(float x, float z, float theta, float& _min, float& _max)
{
    float l = sqrtf(x * x + z * z);
    float pxr = x + z * tan(theta) * l, pzr = z - x * tan(theta) * l;
    float pxl = x - z * tan(theta) * l, pzl = z + x * tan(theta) * l;
    if (pzl <= 0.00001f)
        _min = pxl > 0.0f ? FLT_MAX : -FLT_MAX;
    else
        _min = pxl / pzl;
    if (pzr <= 0.00001f)
        _max = pxr > 0.0f ? FLT_MAX : -FLT_MAX;
    else
        _max = pxr / pzr;
}

__device__ void dir_extents(int side, int N, vec3f v, float theta, int &_xmin, int& _xmax, int& _ymin, int& _ymax)
{
    vec3f c = dir_to_side(side, v); // remap to (x,y,z) where side is at z = 1

    if (theta < 0.785398f) // PI/4
    {
        float xmin, xmax, ymin, ymax;
        extents_1d(c.x, c.z, theta, xmin, xmax);
        extents_1d(c.y, c.z, theta, ymin, ymax);

        if (xmin > 1.0f || xmax < -1.0f || ymin > 1.0f || ymax < -1.0f)
        {
            _xmin = -1; _xmax = -1; _ymin = -1; _ymax = -1; // Bad aabb
        }
        else
        {
            _xmin = (int)min(max((xmin + 1.0f) * (0.5f * (float)N), 0.0f), (float)(N - 1));
            _xmax = (int)min(max((xmax + 1.0f) * (0.5f * (float)N), 0.0f), (float)(N - 1));
            _ymin = (int)min(max((ymin + 1.0f) * (0.5f * (float)N), 0.0f), (float)(N - 1));
            _ymax = (int)min(max((ymax + 1.0f) * (0.5f * (float)N), 0.0f), (float)(N - 1));
        }
    }
    else
    {
            _xmin = 0.0f;
            _xmax = (float)(N-1);
            _ymin = 0.0f;
            _ymax = (float)(N-1);
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// Diffuse kernel
__global__ void DiffuseCubemapFwdKernel(DiffuseCubemapKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    int Npx = p.cubemap.dims[1];
    vec3f N = cube_to_dir(px, py, pz, Npx);

    vec3f col(0);

    for (int s = 0; s < p.cubemap.dims[0]; ++s)
    {
        for (int y = 0; y < Npx; ++y)
        {
            for (int x = 0; x < Npx; ++x)
            {
                vec3f L = cube_to_dir(x, y, s, Npx);
                float costheta = min(max(dot(N, L), 0.0f), 0.999f);
                float w = costheta * pixel_area(x, y, Npx) / 3.141592f; // pi = area of positive hemisphere
                col += p.cubemap.fetch3(x, y, s) * w;
            }
        }
    }

    p.out.store(px, py, pz, col);
}

__global__ void DiffuseCubemapBwdKernel(DiffuseCubemapKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    int Npx = p.cubemap.dims[1];
    vec3f N = cube_to_dir(px, py, pz, Npx);
    vec3f grad = p.out.fetch3(px, py, pz);

    for (int s = 0; s < p.cubemap.dims[0]; ++s)
    {
        for (int y = 0; y < Npx; ++y)
        {
            for (int x = 0; x < Npx; ++x)
            {
                vec3f L = cube_to_dir(x, y, s, Npx);
                float costheta = min(max(dot(N, L), 0.0f), 0.999f);
                float w = costheta * pixel_area(x, y, Npx) / 3.141592f; // pi = area of positive hemisphere
                atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 0), grad.x * w);
                atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 1), grad.y * w);
                atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 2), grad.z * w);
            }
        }
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// GGX splitsum kernel 

__device__ inline float ndfGGX(const float alphaSqr, const float cosTheta)
{
    float _cosTheta = clamp(cosTheta, 0.0, 1.0f);
    float d = (_cosTheta * alphaSqr - _cosTheta) * _cosTheta + 1.0f;
    return alphaSqr / (d * d * M_PI);
}

__global__ void SpecularBoundsKernel(SpecularBoundsKernelParams p)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    int Npx = p.gridSize.x;
    vec3f VNR = cube_to_dir(px, py, pz, Npx);

    const int TILE_SIZE = 16;

    // Brute force entire cubemap and compute bounds for the cone
    for (int s = 0; s < p.gridSize.z; ++s)
    {
        // Assume empty BBox 
        int _min_x = p.gridSize.x - 1, _max_x = 0;
        int _min_y = p.gridSize.y - 1, _max_y = 0;
        
        // For each (8x8) tile
        for (int tx = 0; tx < (p.gridSize.x + TILE_SIZE - 1) / TILE_SIZE; tx++)
        {
            for (int ty = 0; ty < (p.gridSize.y + TILE_SIZE - 1) / TILE_SIZE; ty++)
            {
                // Compute tile extents
                int tsx = tx * TILE_SIZE, tsy = ty * TILE_SIZE;
                int tex = min((tx + 1) * TILE_SIZE, p.gridSize.x), tey = min((ty + 1) * TILE_SIZE, p.gridSize.y);

                // Use some blunt interval arithmetics to cull tiles
                vec3f L0 = cube_to_dir(tsx, tsy, s, Npx), L1 = cube_to_dir(tex, tsy, s, Npx);
                vec3f L2 = cube_to_dir(tsx, tey, s, Npx), L3 = cube_to_dir(tex, tey, s, Npx);
                
                float minx = min(min(L0.x, L1.x), min(L2.x, L3.x)), maxx = max(max(L0.x, L1.x), max(L2.x, L3.x));
                float miny = min(min(L0.y, L1.y), min(L2.y, L3.y)), maxy = max(max(L0.y, L1.y), max(L2.y, L3.y));
                float minz = min(min(L0.z, L1.z), min(L2.z, L3.z)), maxz = max(max(L0.z, L1.z), max(L2.z, L3.z));

                float maxdp = max(minx * VNR.x, maxx * VNR.x) + max(miny * VNR.y, maxy * VNR.y) + max(minz * VNR.z, maxz * VNR.z);
                if (maxdp >= p.costheta_cutoff)
                {
                    // Test all pixels in tile.
                    for (int y = tsy; y < tey; ++y)
                    {
                        for (int x = tsx; x < tex; ++x)
                        {
                            vec3f L = cube_to_dir(x, y, s, Npx);
                            if (dot(L, VNR) >= p.costheta_cutoff)
                            {
                                _min_x = min(_min_x, x);
                                _max_x = max(_max_x, x);
                                _min_y = min(_min_y, y);
                                _max_y = max(_max_y, y);
                            }
                        }
                    }
                }
            }
        }
        p.out.store(p.out._nhwcIndex(pz, py, px, s * 4 + 0), _min_x);
        p.out.store(p.out._nhwcIndex(pz, py, px, s * 4 + 1), _max_x);
        p.out.store(p.out._nhwcIndex(pz, py, px, s * 4 + 2), _min_y);
        p.out.store(p.out._nhwcIndex(pz, py, px, s * 4 + 3), _max_y);
    }
}

__global__ void SpecularCubemapFwdKernel(SpecularCubemapKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    int Npx = p.cubemap.dims[1];
    vec3f VNR = cube_to_dir(px, py, pz, Npx);

    float alpha = p.roughness * p.roughness;
    float alphaSqr = alpha * alpha;

    float wsum = 0.0f;
    vec3f col(0);
    for (int s = 0; s < p.cubemap.dims[0]; ++s)
    {
        int xmin, xmax, ymin, ymax;
        xmin = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 0));
        xmax = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 1));
        ymin = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 2));
        ymax = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 3));

        if (xmin <= xmax)
        {
            for (int y = ymin; y <= ymax; ++y)
            {
                for (int x = xmin; x <= xmax; ++x)
                {
                    vec3f L = cube_to_dir(x, y, s, Npx);
                    if (dot(L, VNR) >= p.costheta_cutoff)
                    {
                        vec3f H = safeNormalize(L + VNR);

                        float wiDotN = max(dot(L, VNR), 0.0f);
                        float VNRDotH = max(dot(VNR, H), 0.0f);

                        float w = wiDotN * ndfGGX(alphaSqr, VNRDotH) * pixel_area(x, y, Npx) / 4.0f;
                        col += p.cubemap.fetch3(x, y, s) * w;
                        wsum += w;
                    }
                }
            }
        }
    }

    p.out.store(p.out._nhwcIndex(pz, py, px, 0), col.x);
    p.out.store(p.out._nhwcIndex(pz, py, px, 1), col.y);
    p.out.store(p.out._nhwcIndex(pz, py, px, 2), col.z);
    p.out.store(p.out._nhwcIndex(pz, py, px, 3), wsum);
}

__global__ void SpecularCubemapBwdKernel(SpecularCubemapKernelParams p)
{
    // Calculate pixel position.
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int pz = blockIdx.z;
    if (px >= p.gridSize.x || py >= p.gridSize.y || pz >= p.gridSize.z)
        return;

    int Npx = p.cubemap.dims[1];
    vec3f VNR = cube_to_dir(px, py, pz, Npx);

    vec3f grad = p.out.fetch3(px, py, pz);

    float alpha = p.roughness * p.roughness;
    float alphaSqr = alpha * alpha;

    vec3f col(0);
    for (int s = 0; s < p.cubemap.dims[0]; ++s)
    {
        int xmin, xmax, ymin, ymax;
        xmin = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 0));
        xmax = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 1));
        ymin = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 2));
        ymax = (int)p.bounds.fetch(p.bounds._nhwcIndex(pz, py, px, s * 4 + 3));

        if (xmin <= xmax)
        {
            for (int y = ymin; y <= ymax; ++y)
            {
                for (int x = xmin; x <= xmax; ++x)
                {
                    vec3f L = cube_to_dir(x, y, s, Npx);
                    if (dot(L, VNR) >= p.costheta_cutoff)
                    {
                        vec3f H = safeNormalize(L + VNR);

                        float wiDotN = max(dot(L, VNR), 0.0f);
                        float VNRDotH = max(dot(VNR, H), 0.0f);

                        float w = wiDotN * ndfGGX(alphaSqr, VNRDotH) * pixel_area(x, y, Npx) / 4.0f;

                        atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 0), grad.x * w);
                        atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 1), grad.y * w);
                        atomicAdd((float*)p.cubemap.d_val + p.cubemap.nhwcIndexContinuous(s, y, x, 2), grad.z * w);
                    }
                }
            }
        }
    }
}
